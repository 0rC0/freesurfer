#include "hip/hip_runtime.h"
#include "visitcountertetrahedralmeshcudaimpl.hpp"

#include "cudautils.hpp"

#include "simplesharedtetrahedron.hpp"

const unsigned int nDims = 3;
const unsigned int nVertices = 4;

// Largely copied from visitcountersimplecudaimpl.cu
// TODO Refactor common code
template<typename T,typename Internal>
__global__
void TetrahedralMeshVisitCounterKernel( kvl::cuda::Image_GPU<int,3,unsigned short> output,
					const kvl::cuda::TetrahedralMesh_GPU<T,unsigned long> mesh ) {
  const size_t iTet = blockIdx.x + (gridDim.x * blockIdx.y);
  
  // Check if this block has an assigned tetrahedron
  if( iTet >= mesh.GetTetrahedraCount() ) {
    return;
  }

  // Load the tetrahedron and determine bounding box
  __shared__ T tetrahedron[nVertices][nDims];
  __shared__ unsigned short min[nDims], max[nDims];
  __shared__ T M[nDims][nDims];
  SimpleSharedTetrahedron<T,Internal> tet(tetrahedron, M);

  tet.LoadAndBoundingBox( mesh, iTet, min, max );

  tet.ComputeBarycentricTransform();

  // Figure out how to cover the bounding box with the current thread block
  // We assume that each thread block is strictly 2D

  // Divide the bounding box into blocks equal to the blockDim
  for( unsigned short iyStart=min[1]; iyStart<max[1]; iyStart += blockDim.y ) {
    for( unsigned short ixStart=min[0]; ixStart<max[0]; ixStart += blockDim.x ) {
      const unsigned short ix = ixStart + threadIdx.x;
      const unsigned short iy = iyStart + threadIdx.y;

      // Could probably do this test a little better
      if( output.PointInRange(0,iy,ix) ) {

	for( unsigned short iz=min[2]; iz<max[2]; iz++ ) {
	  bool inside = tet.PointInside(ix,iy,iz);
	  
	  if( inside ) {
	    atomicAdd(&output(iz,iy,ix),1);
	  }
	}
      }
    }
  }
}



namespace kvl {
  namespace cuda {
    
    void RunVisitCounterTetrahedralMeshCUDA( CudaImage<int,3,unsigned short>& d_output,
					     const CudaTetrahedralMesh<double,unsigned long>& ctm ) {
      const unsigned int nBlockx = 1024;

      const size_t nTetrahedra = ctm.GetTetrahedraCount();

      const unsigned int nThreadsx = GetBlockSize( d_output.ElementCount(), nTetrahedra );
      const unsigned int nThreadsy = GetBlockSize( d_output.ElementCount(), nTetrahedra );
      const unsigned int nThreadsz = 1;

      dim3 grid, threads;

      
      if( nTetrahedra > nBlockx ) {
	grid.x = nBlockx;
	grid.y = (nTetrahedra / grid.x)+1;
	if( (grid.y * grid.x) < nTetrahedra ) {
	  grid.y++;
	}
      } else {
	grid.x = nTetrahedra;
	grid.y = 1;
      }

      threads.x = nThreadsx;
      threads.y = nThreadsy;
      threads.z = nThreadsz;
      
      // Run the kernel
      auto err = hipGetLastError();
      if( hipSuccess != err ) {
	throw CUDAException(err);
      }
      TetrahedralMeshVisitCounterKernel<double,double><<<grid,threads>>>( d_output.getArg(), ctm.getArg() );
      err = hipDeviceSynchronize();
      if( hipSuccess != err ) {
	throw CUDAException(err);
      }
    } 
  }
}

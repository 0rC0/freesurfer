/**
 * @file  mriframegpu.cu
 * @brief Holds MRI frame template for the GPU
 *
 * Holds an MRI frame template type for the GPU
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2010/01/19 16:55:33 $
 *    $Revision: 1.1 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 * Bug reports: analysis-bugs@nmr.mgh.harvard.edu
 *
 */


#include "mriframegpu.hpp"

// ====================================================


template<> int GetAsMRItype<unsigned char>( const unsigned char tmp ) {
  return( MRI_UCHAR );
}



// ====================================================

template<>
void CopyMRIrowToContiguous<unsigned char>( const MRI* src, unsigned char* h_slab,
					    const unsigned int iy,
					    const unsigned int iz,
					    const unsigned int iFrame ) {
  // Sanity checks
  if( src->type != GetAsMRItype( h_slab[0] ) ) {
    std::cerr << __PRETTY_FUNCTION__ << ": Type mismatch" << std::endl;
    exit( EXIT_FAILURE );
  }

  if( iFrame >= src->nframes ) {
    std::cerr << __PRETTY_FUNCTION__ << ": Frame out of range" << std::endl;
    exit( EXIT_FAILURE );
  }

  // Do the copy
  memcpy( h_slab,
	  &MRIseq_vox( src, 0, iy, iz, iFrame ),
	  src->width*sizeof(unsigned char) );
}


// ======================================================


static void NullFunction( void ) {
  MRIframeGPU<unsigned char> tstFrame;

  tstFrame.cpuDims.x = 0;
}

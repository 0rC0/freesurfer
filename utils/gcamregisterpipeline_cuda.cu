
#include <hip/hip_runtime.h>
/**
 * @file  gcamregisterpipeline_cuda.cu
 * @brief Implementation of GCAMregisterPipeline for the GPU
 *
 * Reference:
  * "Whole Brain Segmentation: Automated Labeling of Neuroanatomical
  * Structures in the Human Brain", Fischl et al.
  * (2002) Neuron, 33:341-355.
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: zkaufman $
 *    $Date: 2016/02/04 20:23:05 $
 *    $Revision: 1.5 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */


#ifdef GCAMORPH_ON_GPU

#include "macros.h"
#include "error.h"

#include "gcamorph.h"

#include "chronometer.hpp"

#include "mriframegpu.hpp"
#include "gcamorphgpu.hpp"


#include "gcamorphenergy.hpp"
#include "gcamregisterlevel_cuda.hpp"


// ========================================================================

template<typename T, typename U>
float RegisterPipeline( GPU::Classes::GCAmorphGPU& gcam,
			const GPU::Classes::MRIframeGPU<T>& mri,
			const GPU::Classes::MRIframeGPU<U>& mri_smooth,
			GCA_MORPH_PARMS *parms,
			double *last_rms,
			int *level_steps,
			int i )
{
  GPU::Algorithms::GCAmorphEnergy gcamEnergy;

  *last_rms = gcamEnergy.ComputeRMS( gcam, mri, parms );
  if( i==0 )
  {
    parms->start_rms = *last_rms;
  }
  *level_steps = parms->start_t;
  RegisterLevel( gcam, mri, mri_smooth, parms);

  return gcamEnergy.ComputeRMS( gcam, mri, parms );
}


// ================

template<typename T, typename U>
float
gcamRPfinalDispatch( GCA_MORPH *gcam,
                     MRI *mri,
                     MRI *mri_smooth,
                     GCA_MORPH_PARMS *parms,
                     double *last_rms,
                     int *level_steps,
                     int i )
{
  GPU::Classes::GCAmorphGPU myGCAM;
  GPU::Classes::MRIframeGPU<T> myMRI;
  GPU::Classes::MRIframeGPU<U> myMRIsmooth;
  float result;

  // Handle the MRIs
  myMRI.Allocate( mri );
  myMRI.Send( mri, 0 );

  myMRIsmooth.Allocate( mri_smooth );
  myMRIsmooth.Send( mri_smooth, 0 );

  // Put the GCAM on the GPU
  myGCAM.CheckIntegrity(); // Shouldn't be necessary....
  myGCAM.SendAll( gcam );

  // Run the computation
  result = RegisterPipeline( myGCAM, myMRI, myMRIsmooth, parms,
			     last_rms, level_steps, i);

  // Retrieve results
  myGCAM.RecvAll( gcam );

  return result;
}


// -----------


template<typename T>
float
gcamRPsmoothDispatch(  GCA_MORPH *gcam,
                       MRI *mri,
                       MRI *mri_smooth,
                       GCA_MORPH_PARMS *parms,
                       double *last_rms,
                       int *level_steps,
                       int i )
{
  float result;

  switch( mri_smooth->type )
  {

  case MRI_UCHAR:
    result = gcamRPfinalDispatch<T,unsigned char>( gcam, mri, mri_smooth, parms,
						   last_rms, level_steps, i );
    break;
  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised type for mri_smooth "
              << mri_smooth->type << std::endl;
    abort();
  }

  return result;
}


// -------------------

float GCAMregisterPipelineAndComputeRMSGPU( GCA_MORPH *gcam,
					    MRI *mri,
					    MRI *mri_smooth,
					    GCA_MORPH_PARMS *parms,
					    double *last_rms,
					    int *level_steps,
					    int i )
{
  float result;

  switch( mri->type )
  {

  case MRI_UCHAR:
    result = gcamRPsmoothDispatch<unsigned char>( gcam, mri, mri_smooth, parms,
						  last_rms, level_steps, i );
    break;

  case MRI_FLOAT:
    result = gcamRPsmoothDispatch<float>( gcam, mri, mri_smooth, parms,
					  last_rms, level_steps, i );
    break;
    
  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised type for mri "
              << mri->type << std::endl;
    abort();
  }

  return result;
}


#endif

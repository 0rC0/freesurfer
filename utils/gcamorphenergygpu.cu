#include "hip/hip_runtime.h"
/**
 * @file  gcamorphenergygpu.cu
 * @brief Holds routines to compute GCAmorph energies on the GPU
 *
 * 
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2010/03/17 19:03:33 $
 *    $Revision: 1.9 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include <thrust/device_new_allocator.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#include "cma.h"

#include "chronometer.hpp"

#include "mriframegpu.hpp"
#include "gcamorphgpu.hpp"

// Stolen from gcamorph.c
#define MIN_STD 2
#define MIN_VAR (MIN_STD*MIN_STD)


//! Texture reference for an unsigned char mri
texture<unsigned char, 3, hipReadModeNormalizedFloat> dt_mri_uchar;


// ==============================================================

namespace GPU {
  namespace Algorithms {


    const unsigned int kGCAmorphLLEkernelSize = 16;

    //! Templated texture fetch
    template<typename T>
    __device__ float FetchMRIVoxel( const float3 r ) {
      /*!
	Does look ups into the textures.
	Recall that the textures are configured to return
	normalised floats (except for the float texture!)
	in order to enable linear filtering.
	This in turn requires a conversion from the normalised
	value back to the true value.
	Unspecialised version writes junk
      */
      return( 100000 );
    }

    template<>
    __device__ float FetchMRIVoxel<unsigned char>( const float3 r ) {

      float texVal;
      texVal = tex3D( dt_mri_uchar, r.x+0.5f, r.y+0.5f, r.z+0.5f );
      texVal *= UCHAR_MAX;

      return( texVal );
    }


    __device__
    float GCAmorphDist( const float mean, const float variance,
			const float val ) {
      float v;
      v = val - mean;
      v = v*v;
      v /= variance;

      return( sqrtf( v ) );
    }

    __device__
    bool IsUnknown( const int label ) {
      bool res;

      res = (label==Unknown);
      res = res || (label==255);
      res = res || (label==Bright_Unknown);
      res = res || (label==Dark_Unknown);
      
      return(res);
    }



    __global__
    void ComputeGood( const GPU::Classes::VolumeArgGPU<char> invalid,
		      const GPU::Classes::VolumeArgGPU<int> label,
		      const GPU::Classes::VolumeArgGPU<int> status,
		      GPU::Classes::VolumeArgGPU<char> good ) {

      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;


      // Loop over z slices
      for( unsigned int iz = 0; iz< good.dims.z; iz++ ) {

	// Only compute if ix, iy & iz are inside the bounding box
	if( good.InVolume(ix,iy,iz) ) {
	  good(ix,iy,iz) = 0;
	  
	  // Is it valid?
	  if( invalid(ix,iy,iz) == GCAM_POSITION_INVALID ) {
	    // If not, go to next z slice
	    continue;
	  }
	  
	  // What's the status?
	  if( status(ix,iy,iz) &
	      (GCAM_IGNORE_LIKELIHOOD|GCAM_NEVER_USE_LIKELIHOOD) ) {
	    // Go to next z slice
	    continue;
	  }
	  
	  // Don't use unknowns unless they border known
	  if( IS_UNKNOWN(label(ix,iy,iz)) ) {
	    unsigned int diffLabels = 0;
	    const int myLabel = label(ix,iy,iz);

	    for( unsigned int k=max(0,iz-1);
		 k<=min(invalid.dims.z-1,iz+1);
		 k++ ) {
	      for( unsigned int j=max(0,iy-1);
		   j<=min(invalid.dims.y-1,iy+1);
		   j++ ) {
		for( unsigned int i=max(0,ix-1);
		     i<=min(invalid.dims.x-1,ix+1);
		     i++ ) {
		  if( label(i,j,k) != myLabel ) {
		    diffLabels++;
		  }
		}
	      }
	    }

	    if( diffLabels == 0 ) {
	      // Go to next z slice
	      continue;
	    }
	  }

	  // If we get to here, it's OK
	  good(ix,iy,iz) = 1;
	}
      }
    }



    template<typename T>
    __global__
    void ComputeLLE( const GPU::Classes::VolumeArgGPU<float> rx,
		     const GPU::Classes::VolumeArgGPU<float> ry,
		     const GPU::Classes::VolumeArgGPU<float> rz,
		     const GPU::Classes::VolumeArgGPU<char> good,
		     const GPU::Classes::VolumeArgGPU<float> mean,
		     const GPU::Classes::VolumeArgGPU<float> variance,
		     double* energies ) {
      
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;

      float myEnergy;


      // Loop over z slices
      for( unsigned int iz = 0; iz < rx.dims.z; iz++ ) {

	// Only compute if ix, iy & iz are inside the bounding box
	if( rx.InVolume(ix,iy,iz) ) {

	  const unsigned int iLoc = rx.Index1D( ix, iy, iz );
	  
	  // See if we want to do this pixel
	  if( good(ix,iy,iz) == 0 ) {
	    continue;
	  }

	  float3 r = make_float3( rx( ix, iy, iz ),
				  ry( ix, iy, iz ),
				  rz( ix, iy, iz ) );

	  // Get the MRI value, clamping exterior to 0
	  float mriVal = 0;

	  if( rx.InFuzzyVolume( r, 0.5f ) ) {
	    mriVal = FetchMRIVoxel<T>( r );
	  }
	  
	  // Compute contribution to the energy
	  if( variance(ix,iy,iz) >= 0 ) {
	    // We have a valid variance
	    myEnergy = GCAmorphDist( mean(ix,iy,iz),
				     variance(ix,iy,iz),
				     mriVal );
	    myEnergy += logf( variance(ix,iy,iz) );
	  } else {
	    myEnergy = mriVal*mriVal / MIN_VAR;
	  }

	  energies[iLoc] = myEnergy;
	}
      }
    }


    //! Class to hold GCAMorph energy computations
    class GCAmorphEnergy {
    public:


      //! Implementation of gcamLogLikelihoodEnergy for the GPU
      template<typename T>
      float LogLikelihoodEnergy( const GPU::Classes::GCAmorphGPU& gcam,
				 const GPU::Classes::MRIframeGPU<T>& mri ) {
	/*!
	  This the the host side function for
	  gcamLogLikelihoodEnergy on the GPU.
	  Note that a GCAmorphGPU implicitly only has one input for
	  each location.
	  This means that each covariance is just a variance,
	  and negative values flag
	*/

	// Make sure the GCAM is sane
	gcam.CheckIntegrity();

	// Get the MRI texture in place (must be in CUDA array already)
	this->BindMRI( mri );

	const dim3 gcamDims = gcam.d_rx.GetDims();
	const unsigned int nVoxels = gcamDims.x * gcamDims.y * gcamDims.z;
	// Create a 'flag' array
	GPU::Classes::VolumeGPU<char> d_good;
	d_good.Allocate( gcamDims );

	// Allocate thrust arrays
	std::cout << "Make d_energies double for debug" << std::endl;
	thrust::device_ptr<double> d_energies;
	d_energies = thrust::device_new<double>( nVoxels );
#if 1
	std::cout << __FUNCTION__ << "Zero energies for debug" << std::endl;
	CUDA_SAFE_CALL( hipMemset( thrust::raw_pointer_cast( d_energies ),
				    0,	
				    nVoxels*sizeof(*thrust::raw_pointer_cast( d_energies )) ) );
#endif

	// Get the MRI into a texture
	this->BindMRI( mri );


	// Run the computation
	dim3 grid, threads;
	threads.x = threads.y = kGCAmorphLLEkernelSize;
	threads.z = 1;

	grid = gcam.d_rx.CoverBlocks( kGCAmorphLLEkernelSize );
	grid.z = 1;

	ComputeGood<<<grid,threads>>>( gcam.d_invalid,
				       gcam.d_label,
				       gcam.d_status,
				       d_good );
	CUDA_CHECK_ERROR( "ComputeGood kernel failed!\n" );

	ComputeLLE<T><<<grid,threads>>>
	  ( gcam.d_rx, gcam.d_ry, gcam.d_rz,
	    d_good,
	    gcam.d_mean, gcam.d_variance,
	    thrust::raw_pointer_cast( d_energies ) );
	CUDA_CHECK_ERROR( "ComputeLLE kernel failed!\n" );

#if 0
	for( unsigned int i=0; i<nVoxels; i++ ) {
	  std::cout << i << " " << d_energies[i] << std::endl;
	}
#endif

	// Release the MRI texture
	this->UnbindMRI<T>();

	// Get the sum of the energies
	double energy = thrust::reduce( d_energies, d_energies+nVoxels );

	std::cout << __FUNCTION__
		  << " " << std::setprecision(20) << std::setw(40)
		  << energy << std::endl;

	// Release thrust arrays
	thrust::device_delete( d_energies );

	return( energy );
      }


      //! Dispatch wrapper for LogLikelihoodEnergy
      template<typename T>
      float LLEdispatch( const GCA_MORPH *gcam,
			 const MRI* mri ) {
	
	float energy;

	GPU::Classes::GCAmorphGPU myGCAM;
	myGCAM.SendAll( gcam );

	GPU::Classes::MRIframeGPU<T> myMRI;
	myMRI.Allocate( mri );
	myMRI.Send( mri, 0 );
	myMRI.AllocateArray();
	myMRI.SendArray();
	
	energy = this->LogLikelihoodEnergy( myGCAM, myMRI );

	return( energy );

      }


    private:


      //! Templated texture binding wrapper
      template<typename T>
      void BindMRI( const GPU::Classes::MRIframeGPU<T>& mri ) const {
	std::cerr << __PRETTY_FUNCTION__
		  << ": Unrecognised MRI type" << std::endl;
	exit( EXIT_FAILURE );
      }

      //! Templated texture unbinding
      template<typename T>
      void UnbindMRI( void ) const {
	std::cerr << __PRETTY_FUNCTION__
		  << ": Unrecognised MRI type" << std::endl;
	exit( EXIT_FAILURE );
      }
    };


    template<>
    void GCAmorphEnergy::BindMRI<unsigned char>( const GPU::Classes::MRIframeGPU<unsigned char>& mri ) const {

      dt_mri_uchar.normalized = false;
      dt_mri_uchar.addressMode[0] = hipAddressModeClamp;
      dt_mri_uchar.addressMode[1] = hipAddressModeClamp;
      dt_mri_uchar.addressMode[2] = hipAddressModeClamp;
      dt_mri_uchar.filterMode = hipFilterModeLinear;
      
      CUDA_SAFE_CALL( hipBindTextureToArray( dt_mri_uchar,
					      mri.GetArray() ) );
    }

    template<>
    void GCAmorphEnergy::UnbindMRI<unsigned char>( void ) const {
      CUDA_SAFE_CALL( hipUnbindTexture( dt_mri_uchar ) );
    }
    
  }
}










static GPU::Algorithms::GCAmorphEnergy myEnergy;


//! Wrapper around GPU class
float gcamLogLikelihoodEnergyGPU( const GCA_MORPH *gcam,
				  const MRI* mri ) {
  
  float energy;

  switch( mri->type ) {
  
  case MRI_UCHAR:
    energy = myEnergy.LLEdispatch<unsigned char>( gcam, mri );
    break;


  default:
    std::cerr << __FUNCTION__
	      << ": Unrecognised MRI type" << std::endl;
    exit( EXIT_FAILURE );
  }

  return( energy );

}

/**
 * @file  gcamorphenergygpu.cu
 * @brief Holds routines to compute GCAmorph energies on the GPU
 *
 * 
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2010/03/05 18:48:47 $
 *    $Revision: 1.1 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include "chronometer.hpp"

#include "gcamorphgpu.hpp"




// ==============================================================

namespace GPU {
  namespace Classes {



  }
}

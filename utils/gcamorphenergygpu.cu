#include "hip/hip_runtime.h"
/**
 * @file  gcamorphenergygpu.cu
 * @brief Holds routines to compute GCAmorph energies on the GPU
 *
 * 
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2010/03/16 15:46:17 $
 *    $Revision: 1.6 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include <thrust/device_new_allocator.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

#include "cma.h"

#include "chronometer.hpp"

#include "mriframegpu.hpp"
#include "gcamorphgpu.hpp"

// Stolen from gcamorph.c
#define MIN_STD 2
#define MIN_VAR (MIN_STD*MIN_STD)


//! Texture reference for an unsigned char mri
texture<unsigned char, 3, hipReadModeNormalizedFloat> dt_mri_uchar;


// ==============================================================

namespace GPU {
  namespace Algorithms {


    const unsigned int kGCAmorphLLEkernelSize = 16;

    //! Templated texture fetch
    template<typename T>
    __device__ float FetchMRIVoxel( const float3 r ) {
      /*!
	Does look ups into the textures.
	Recall that the textures are configured to return
	normalised floats (except for the float texture!)
	in order to enable linear filtering.
	This in turn requires a conversion from the normalised
	value back to the true value.
	Unspecialised version writes junk
      */
      return( 100000 );
    }

    template<>
    __device__ float FetchMRIVoxel<unsigned char>( const float3 r ) {

      float texVal;
      texVal = tex3D( dt_mri_uchar, r.x+0.5f, r.y+0.5f, r.z+0.5f );
      texVal *= UCHAR_MAX;

      return( texVal );
    }


    __device__
    float GCAmorphDist( const float mean, const float variance,
			const float val ) {
      float v;
      v = val - mean;
      v = v*v;
      v /= variance;

      return( sqrtf( v ) );
    }

    __device__
    bool IsUnknown( const int label ) {
      bool res;

      res = (label==Unknown);
      res = res || (label==255);
      res = res || (label==Bright_Unknown);
      res = res || (label==Dark_Unknown);
      
      return(res);
    }
#if 0
    template<typename T>
    __global__
    void ComputeLLE( const GPU::Classes::VolumeArgGPU<float3> r,
		     const GPU::Classes::VolumeArgGPU<char> invalid,
		     const GPU::Classes::VolumeArgGPU<int> status,
		     const GPU::Classes::VolumeArgGPU<int> label,
		     const GPU::Classes::VolumeArgGPU<float> mean,
		     const GPU::Classes::VolumeArgGPU<float> variance,
		     float* energies ) {
      
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.x * blockDim.x );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;

      __shared__ int labelCache[3][kGCAmorphLLEkernelSize+2][kGCAmorphLLEkernelSize+2];

      float myEnergy;

      // Begin loading up the labelCache
      int myLabel;

      if( label.InVolume(ix,iy,0) ) {
	myLabel = label(ix,iy,0);
      } else {
	myLabel = Unknown;
      }
      labelCache[0][1+threadIdx.y][1+threadIdx.x] = myLabel;
      labelCache[1][1+threadIdx.y][1+threadIdx.x] = myLabel;

      // Loop over z slices
      for( unsigned int iz = 0; iz< r.dims.z; iz++ ) {

	// We need to do all the other checks done by the CPU routine

	// Fill in the 'above' slab of the label cache
	if( label.InVolume(ix,iy,iz+1) ) {
	  myLabel = label(ix,iy,iz+1);
	} else {
	  myLabel = Unknown;
	}
	labelCache[2][1+threadIdx.y][1+threadIdx.x] = myLabel;
	// Fill in the edges
	if( threadIdx.x < kGCAmorphLLEkernelSize ) {
	  if( label.InVolume(bx-1,by+threadIdx.x,iz) ) {
	    myLabel = label(bx-1,by+threadIdx.x,iz);
	  } else {
	    myLabel = Unknown;
	  }
	  labelCache[1][threadIdx.x][0] = myLabel;

	  if( label.InVolume(bx+kGCAmorphLLEkernelSize,by+threadIdx.x,iz) ) {
	    myLabel = label(bx+kGCAmorphLLEkernelSize,by+threadIdx.x,iz);
	  } else {
	    myLabel = Unknown;
	  }
	  labelCache[1][threadIdx.x][kGCAmorphLLEkernelSize+1] = myLabel;

	  if( label.InVolume(ix,by-1,iz) ) {
	    myLabel = label(ix,by-1,iz);
	  } else {
	    myLabel = Unknown;
	  }
	  labelCache[1][0][1+threadIdx.x] = myLabel;

	  if( label.InVolume(ix,by+kGCAmorphLLEkernelSize,iz) ) {
	    myLabel = label(ix,by+kGCAmorphLLEkernelSize,iz);
	  } else {
	    myLabel = Unknown;
	  }
	  labelCache[1][kGCAmorphLLEkernelSize+1][1+threadIdx.x] = myLabel;

	  // Still need the corners (and above, too!)
	}


	__syncthreads();

	// Only compute if ix, iy & iz are inside the bounding box
	if( r.InVolume(ix,iy,iz) ) {

	  if( invalid(ix,iy,iz) == GCAM_POSITION_INVALID ) {
	    continue;
	  }

	  if( status(ix,iy,iz) &
	      (GCAM_IGNORE_LIKELIHOOD|GCAM_NEVER_USE_LIKELIHOOD) ) {
	    continue ;
	  }

	  // Get the MRI value, clamping exterior to 0
	  float mriVal = 0;
	  if( r.InFuzzyVolume( r(ix,iy,iz), 0.5f ) ) {
	    mriVal = FetchMRIVoxel<T>( r(ix,iy,iz) );
	  }
	  
	  // Compute contribution to the energy
	  if( variance(ix,iy,iz) >= 0 ) {
	    // We have a valid variance
	    myEnergy = GCAmorphDist( mean(ix,iy,iz),
				     variance(ix,iy,iz),
				     mriVal );
	    myEnergy += logf( variance(ix,iy,iz) );
	  } else {
	    myEnergy = mriVal*mriVal / MIN_VAR;
	  }

	  const unsigned int iLoc = r.Index1D( ix, iy, iz );
	  energies[iLoc] = myEnergy;
	}
      }
    }

#endif


    //! Class to hold GCAMorph energy computations
    class GCAmorphEnergy {
    public:


      //! Implementation of gcamLogLikelihoodEnergy for the GPU
      template<typename T>
      float LogLikelihoodEnergy( const GPU::Classes::GCAmorphGPU& gcam,
				 const GPU::Classes::MRIframeGPU<T>& mri ) {
	/*!
	  This the the host side function for
	  gcamLogLikelihoodEnergy on the GPU.
	  Note that a GCAmorphGPU implicitly only has one input for
	  each location.
	  This means that each covariance is just a variance,
	  and negative values flag
	*/

	// Make sure the GCAM is sane
	gcam.CheckIntegrity();

	// Get the MRI texture in place (must be in CUDA array already)
	this->BindMRI( mri );

	const dim3 gcamDims = gcam.d_rx.GetDims();
	const unsigned int nVoxels = gcamDims.x * gcamDims.y * gcamDims.z;

	// Allocate thrust arrays
	thrust::device_ptr<float> d_energies;
	d_energies = thrust::device_new<float>( nVoxels );

	// Get the MRI into a texture
	this->BindMRI( mri );


	// Run the computation
	dim3 grid, threads;
	threads.x = threads.y = kGCAmorphLLEkernelSize;
	threads.z = 1;

	grid = gcam.d_rx.CoverBlocks( kGCAmorphLLEkernelSize );
	grid.z = 1;

#if 0
	ComputeLLE<T><<<grid,threads>>>
	  ( gcam.d_r, gcam.d_invalid, gcam.d_status,
	    gcam.d_label, gcam.d_mean, gcam.d_variance,
	    thrust::raw_pointer_cast( d_energies ) );
	CUDA_CHECK_ERROR( "ComputeLLE kernel failed!\n" );
#endif


	// Release the MRI texture
	this->UnbindMRI<T>();

	// Get the sum of the energies
	float energy = thrust::reduce( d_energies, d_energies+nVoxels );


	// Release thrust arrays
	thrust::device_delete( d_energies );

	return( energy );
      }


      //! Dispatch wrapper for LogLikelihoodEnergy
      template<typename T>
      float LLEdispatch( const GCA_MORPH *gcam,
			 const MRI* mri ) {
	
	float energy;

	GPU::Classes::GCAmorphGPU myGCAM;
	myGCAM.SendAll( gcam );

	GPU::Classes::MRIframeGPU<T> myMRI;
	myMRI.Send( mri, 0 );
	myMRI.AllocateArray();
	myMRI.SendArray();
	
	energy = this->LogLikelihoodEnergy( myGCAM, myMRI );

	return( energy );

      }


    private:


      //! Templated texture binding wrapper
      template<typename T>
      void BindMRI( const GPU::Classes::MRIframeGPU<T>& mri ) const {
	std::cerr << __PRETTY_FUNCTION__
		  << ": Unrecognised MRI type" << std::endl;
	exit( EXIT_FAILURE );
      }

      //! Templated texture unbinding
      template<typename T>
      void UnbindMRI( void ) const {
	std::cerr << __PRETTY_FUNCTION__
		  << ": Unrecognised MRI type" << std::endl;
	exit( EXIT_FAILURE );
      }
    };


    template<>
    void GCAmorphEnergy::BindMRI<unsigned char>( const GPU::Classes::MRIframeGPU<unsigned char>& mri ) const {

      dt_mri_uchar.normalized = false;
      dt_mri_uchar.addressMode[0] = hipAddressModeClamp;
      dt_mri_uchar.addressMode[1] = hipAddressModeClamp;
      dt_mri_uchar.addressMode[2] = hipAddressModeClamp;
      dt_mri_uchar.filterMode = hipFilterModeLinear;
      
      CUDA_SAFE_CALL( hipBindTextureToArray( dt_mri_uchar,
					      mri.GetArray() ) );
    }

    template<>
    void GCAmorphEnergy::UnbindMRI<unsigned char>( void ) const {
      CUDA_SAFE_CALL( hipUnbindTexture( dt_mri_uchar ) );
    }
    
  }
}










static GPU::Algorithms::GCAmorphEnergy myEnergy;


//! Wrapper around GPU class
float gcamLogLikelihoodEnergyGPU( const GCA_MORPH *gcam,
				  const MRI* mri ) {
  
  float energy;

  switch( mri->type ) {
  
  case MRI_UCHAR:
    energy = myEnergy.LLEdispatch<unsigned char>( gcam, mri );
    break;


  default:
    std::cerr << __FUNCTION__
	      << ": Unrecognised MRI type" << std::endl;
    exit( EXIT_FAILURE );
  }

  return( energy );

}

#include "hip/hip_runtime.h"
/**
 * @file  vecvolgpu.cu
 * @brief Holds datatype for float vector volume data on the GPU
 *
 * Holds a datatype for vector volume data on the GPU.
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2011/02/15 14:32:00 $
 *    $Revision: 1.3 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include "cudacheck.h"
#include "cudatypeutils.hpp"

#include "vecvolgpu.hpp"

namespace GPU {

  namespace Classes {

    // ==========================================================
    // Memory management

    void VecVolGPU::Allocate( const dim3 myDims ) {
      
      // Check if we can re-use current memory
      if( myDims == this->dims ) {
        return;
      }

      this->Release();

      this->dims = myDims;

      hipExtent tmpExtent = ExtentFromDims( this->dims );
      tmpExtent.width *= sizeof(float);

      // Allocate the memory
      CUDA_SAFE_CALL( hipMalloc3D( &(this->d_x), tmpExtent ) );
      CUDA_SAFE_CALL( hipMalloc3D( &(this->d_y), tmpExtent ) );
      CUDA_SAFE_CALL( hipMalloc3D( &(this->d_z), tmpExtent ) );
    }

    // ----

    void VecVolGPU::Release( void ) {
      if( this->d_x.ptr != NULL ) {
        CUDA_SAFE_CALL( hipFree( this->d_x.ptr ) );
        CUDA_SAFE_CALL( hipFree( this->d_y.ptr ) );
        CUDA_SAFE_CALL( hipFree( this->d_z.ptr ) );
        this->dims = make_uint3(0,0,0);
        this->d_x = make_hipPitchedPtr(NULL,0,0,0);
        this->d_y = make_hipPitchedPtr(NULL,0,0,0);
        this->d_z = make_hipPitchedPtr(NULL,0,0,0);
      }
    }


    // ----


    size_t VecVolGPU::BufferSize( void ) const {
      size_t nElements;

      nElements = this->dims.x * this->dims.y * this->dims.z;

      return( nElements*sizeof(float) );
    }

    
    // ----

    
    void VecVolGPU::AllocateHostBuffers( float** h_x,
                                         float** h_y,
                                         float** h_z ) const {
      CUDA_SAFE_CALL( hipHostAlloc( h_x,
                                     this->BufferSize(),
                                     hipHostMallocDefault ) );
      CUDA_SAFE_CALL( hipHostAlloc( h_y,
                                     this->BufferSize(),
                                     hipHostMallocDefault ) );
      CUDA_SAFE_CALL( hipHostAlloc( h_z,
                                     this->BufferSize(),
                                     hipHostMallocDefault ) );
      
    }


    // ==========================================================
    // Data transfer

    void VecVolGPU::SendBuffers( const float* const h_x,
                                 const float* const h_y,
                                 const float* const h_z ) {

      hipMemcpy3DParms copyParams = {0};

      copyParams.srcPtr = make_hipPitchedPtr( (void*)h_x,
					       this->dims.x*sizeof(float),
					       this->dims.x,
					       this->dims.y );
      copyParams.dstPtr = this->d_x;
      copyParams.extent = ExtentFromDims( this->dims );
      copyParams.extent.width *= sizeof(float);
      copyParams.kind = hipMemcpyHostToDevice;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );


      copyParams.srcPtr = make_hipPitchedPtr( (void*)h_y,
					       this->dims.x*sizeof(float),
					       this->dims.x,
					       this->dims.y );
      copyParams.dstPtr = this->d_y;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );

      
      copyParams.srcPtr = make_hipPitchedPtr( (void*)h_z,
					       this->dims.x*sizeof(float),
					       this->dims.x,
					       this->dims.y );
      copyParams.dstPtr = this->d_z;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );
    }

    // ----

    void VecVolGPU::RecvBuffers( float* const h_x,
                                 float* const h_y,
                                 float* const h_z ) const {

      hipMemcpy3DParms cpyPrms = {0};
      cpyPrms.srcPtr = this->d_x;
      cpyPrms.dstPtr = make_hipPitchedPtr( (void*)h_x,
					    this->dims.x*sizeof(float),
					    this->dims.x,
					    this->dims.y );
      cpyPrms.extent = ExtentFromDims( this->dims );
      cpyPrms.extent.width *= sizeof(float);
      cpyPrms.kind = hipMemcpyDeviceToHost;
      CUDA_SAFE_CALL( hipMemcpy3D( &cpyPrms ) );

      cpyPrms.srcPtr = this->d_y;
      cpyPrms.dstPtr = make_hipPitchedPtr( (void*)h_y,
					    this->dims.x*sizeof(float),
					    this->dims.x,
					    this->dims.y );
      CUDA_SAFE_CALL( hipMemcpy3D( &cpyPrms ) );

      cpyPrms.srcPtr = this->d_z;
      cpyPrms.dstPtr = make_hipPitchedPtr( (void*)h_z,
					    this->dims.x*sizeof(float),
					    this->dims.x,
					    this->dims.y );
      CUDA_SAFE_CALL( hipMemcpy3D( &cpyPrms ) );

    }


    // ----
#ifdef GCAMORPH_ON_GPU
    void VecVolGPU::CopyFromGCAmorphGPU( const GCAmorphGPU& src ) {
      /*!
        Copies the d_rx, d_ry and d_rz fields from a GCAmorphGPU
        into this volume.
        Really, the GCAmorphGPU should include this class
      */
      this->Allocate( src.d_rx.GetDims() );

      hipMemcpy3DParms copyParams = {0};

      copyParams.srcPtr = src.d_rx.d_data;
      copyParams.dstPtr = this->d_x;
      copyParams.extent = ExtentFromDims( this->dims );
      copyParams.extent.width *= sizeof(float);
      copyParams.kind = hipMemcpyDeviceToDevice;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );
      
      copyParams.srcPtr = src.d_ry.d_data;
      copyParams.dstPtr = this->d_y;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );

      copyParams.srcPtr = src.d_rz.d_data;
      copyParams.dstPtr = this->d_z;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );
    }


    // ----

    void VecVolGPU::CopyToGCAmorphGPU( GCAmorphGPU& dst ) const {

      // Sanity checks
      dst.CheckIntegrity();

      if( this->GetDims() != dst.d_rx.GetDims() ) {
        std::cerr << __FUNCTION__
                  << ": Volume size mismatch"
                  << std::endl;
        abort();
      }

      // Do the copies
      hipMemcpy3DParms copyParams = {0};
      copyParams.srcPtr = this->d_x;
      copyParams.dstPtr = dst.d_rx.d_data;
      copyParams.extent = ExtentFromDims( this->dims );
      copyParams.extent.width *= sizeof(float);
      copyParams.kind = hipMemcpyDeviceToDevice;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );

      copyParams.srcPtr = this->d_y;
      copyParams.dstPtr = dst.d_ry.d_data;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );

      copyParams.srcPtr = this->d_z;
      copyParams.dstPtr = dst.d_rz.d_data;
      CUDA_SAFE_CALL( hipMemcpy3D( &copyParams ) );
    }
#endif

  }
}

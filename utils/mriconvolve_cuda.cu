#include "hip/hip_runtime.h"
/**
 * @file  mriconvolve_cuda.cu
 * @brief Holds MRI convolution functions for the GPU
 *
 * Implements MRI convolutions on the GPU. These routines will be hooked
 * into the higher level routines in the main library.
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2010/01/25 15:26:05 $
 *    $Revision: 1.3 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 * Bug reports: analysis-bugs@nmr.mgh.harvard.edu
 *
 */

#include <cstdlib>
#include <cstdio>

#include <iostream>
#include <iomanip>


extern "C" {
#include "mri.h"
}

//#include "cuPrintf.cu"

#include "chronometer.hpp"
#include "cudacheck.h"


#include "mriframegpu.hpp"


#include "mriconvolve_cuda.h"


//#define SHOW_TIMINGS


/*
  I'd really like the following to be within the namespace, but
  I can't charm nvcc into allowing this. We obviously
  need some more C++ available for CUDA. And probably a real
  linker
*/
    
//! Device constant indicating the number of values in the convolution kernel
__constant__ unsigned int dc_mriconv1d_kernel_nVals;

//! Texture reference to the convolution kernel
texture<float, 1, hipReadModeElementType> dtl_mriconv1d_kernel;


namespace GPU {
  namespace Algorithms {

    const unsigned int kConv1dBlockSize = 16;
    
    // Some timers
    SciGPU::Utilities::Chronometer tMRIconv1dMem, tMRIconv1dMemHost;
    SciGPU::Utilities::Chronometer tMRIconv1dSend, tMRIconv1dRecv;
    SciGPU::Utilities::Chronometer tMRIconv1dCompute;
    SciGPU::Utilities::Chronometer tMRIconv1dTotal;
    
    // =================================================
    
    //! Array to contain the convolution kernel on the device
    static float* d_mriconv1d_kernel;

    // =================================================
    
    //! Prepares convolution kernel for use
    void MRIconv1d_SendKernel( const float* kernel,
			       const unsigned int nVals ) {
      /*!
	This routine is responsible for preparing the dtl_mriconv1d_kernel
	texture for use.
	The array on the device is padded with an extra zero on each end,
	to save us some explicit boundary condition checks (the texture
	units will handle them).
	@param[in] kernel Array containing the kernel values
	@param[in] nVals The number of values in the kernel
      */

      // Allocate and zero GPU memory
      CUDA_SAFE_CALL( hipMalloc( (void**)&d_mriconv1d_kernel,
				  (2+nVals)*sizeof(float) ) );
      CUDA_SAFE_CALL( hipMemset( d_mriconv1d_kernel,
				  0,
				  (2+nVals)*sizeof(float) ) );

      // Copy the convolution kernel to the GPU
      // Note the extra offset
      CUDA_SAFE_CALL( hipMemcpy( &(d_mriconv1d_kernel[1]),
				  kernel,
				  nVals*sizeof(float),
				  hipMemcpyHostToDevice ) );

      // Copy the size of the texture to device constant memory
      CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "dc_mriconv1d_kernel_nVals"),
					  &nVals,
					  sizeof(unsigned int) ) );

      // ------------------
      // Set up the texture
      
      hipChannelFormatDesc cd_kernel = hipCreateChannelDesc<float>();
      
      // Describe the addressing modes
      dtl_mriconv1d_kernel.normalized = false;
      dtl_mriconv1d_kernel.addressMode[0] = hipAddressModeClamp;
      dtl_mriconv1d_kernel.filterMode = hipFilterModePoint;
      
      // Bind the texture together
      CUDA_SAFE_CALL( hipBindTexture( 0,
				       dtl_mriconv1d_kernel,
				       d_mriconv1d_kernel,
				       cd_kernel,
				       (2+nVals)*sizeof(float) ) );
    }
    
    
    // ----------------

    //! Releases convolution kernel after use
    void MRIconv1d_ReleaseKernel( void ) {
      /*!
	Releases everything on the device associated with the convolution kernel
      */
      CUDA_SAFE_CALL( hipUnbindTexture( dtl_mriconv1d_kernel ) );
      CUDA_SAFE_CALL( hipFree( d_mriconv1d_kernel ) );
      d_mriconv1d_kernel = NULL;
    }


    // ----------------

    //! Device function to perform convolution kernel look ups
    __device__ float GetMRIconvkernel( const int i ) {
      /*!
	Performs the texture look up of the convolution kernel.
	The texture will be cached, and the extra zeros padded
	mean we don't need to worry about look ups which are
	out of range.
      */
      
      // Note the additional offset for the zero padding
      return( tex1Dfetch( dtl_mriconv1d_kernel, i+1.5f ) );
    }


    // =================================================


    //! Function to coerce the convolution result to the desired type
    template<typename T>
    __device__ T GetFinal( const float in ) {
      return( in );
    }
    
    
    template<>
    __device__ unsigned char GetFinal<unsigned char>( const float in ) {
      
      return( static_cast<unsigned char>( rintf( in ) ) );
    }
    
    
    template<>
    __device__ short GetFinal<short>( const float in ) {
      
      return( static_cast<short>( rintf( in ) ) );
    }
    
    
    template<>
    __device__ int GetFinal<int>( const float in ) {
      
      return( static_cast<int>( rintf( in ) ) );
    }
    
    
    template<>
    __device__ long GetFinal<long>( const float in ) {
      
      return( static_cast<long>( rintf( in ) ) );
    }
    
    
    // ==================================================
    
    //! Function to round up the convolution kernel size
    __device__ unsigned int MRIRoundUpConvolutionKernelSize( void ) {
      /*!
	To avoid edge cases, we want to make everything into
	units of kConv1dBlockSize.
	This performs the task for the convolution kernel
	Note that we have an extra divide by two since we're only
	really interested in the half-width of the convolution kernel
      */
      float temp;

      temp = static_cast<float>(dc_mriconv1d_kernel_nVals/2) / kConv1dBlockSize ;

      unsigned int n = static_cast<unsigned int>(ceilf( temp ) );

      return( n * kConv1dBlockSize );
    }




    //! Kernel to convolve in the X direction
    template<typename T, typename U>
    __global__ void MRIConvolveKernelX( const GPU::Classes::MRIframeOnGPU<T> src,
					GPU::Classes::MRIframeOnGPU<U> dst ) {
      /*!
	Kernel to do a convolution in the x direction, using the
	convolution kernel set up in the texture.
	Each block will do one 16x16 (x,y) patch of the MRI.
	The z co-ordinate will be in blockIdx.y
	The (x,y) location of the patch will be derived from blockIdx.x.
	Since the slices array is padded out to multiples of 16 by
	the other MRI slices routines (the host routine checks this), we
	don't have to worry about nasty edge cases
      */
      // Extract some co-ordinates
      const unsigned int by = blockIdx.x / (src.dims.x/kConv1dBlockSize);
      const unsigned int bx = blockIdx.x % (src.dims.x/kConv1dBlockSize);
      const unsigned int iz = blockIdx.y;
      const unsigned int tx = threadIdx.x;
      const unsigned int ty = threadIdx.y;
      
      const unsigned int ixStart = bx * kConv1dBlockSize;
      const unsigned int iyStart = by * kConv1dBlockSize;
      const unsigned int myx = ixStart + tx;
      const unsigned int myy = iyStart + ty;
      // Round up the convolution kernel size
      const unsigned int iConvKernelSize = MRIRoundUpConvolutionKernelSize();
      
      // Accumulator value
      float myVoxel = 0;
      
      // Declared as float because the GPU likes 4 byte datatypes
      __shared__ float srcCache[kConv1dBlockSize][kConv1dBlockSize];
      
      // Loop in the X direction, loading sub-blocks of the src array into the cache
      // Note that xBegin can be negative, so we must cast ixStart and iConvKernelSize
      // to signed integers
      int xFirst = static_cast<int>(ixStart) - static_cast<int>(iConvKernelSize);
      int xLast = ixStart + iConvKernelSize;
      
      for( int xBegin = xFirst; xBegin <= xLast; xBegin += kConv1dBlockSize ) {
	// Load the cache
	srcCache[ty][tx] = src( src.ClampCoord(xBegin+tx,src.dims.x), myy, iz );
	__syncthreads();
	
	// Accumulate
	for( unsigned int i=0; i<kConv1dBlockSize; i++ ) {
	  int convLoc;
	  convLoc = (xBegin-ixStart) - static_cast<int>(tx);
	  convLoc += i;
	  convLoc += (dc_mriconv1d_kernel_nVals/2);
	  myVoxel += srcCache[ty][i] * GetMRIconvkernel( convLoc );
	}
	__syncthreads();
      }
      
      dst(myx, myy, iz) = GetFinal<U>(myVoxel);
      
    }
    


    //! Kernel to convolve in the Y direction
    template<typename T, typename U>
    __global__ void MRIConvolveKernelY( const GPU::Classes::MRIframeOnGPU<T> src,
				     GPU::Classes::MRIframeOnGPU<U> dst ) {
      /*!
	Kernel to do a convolution in the y direction, using the
	convolution kernel set up in the texture.
	Each block will do one 16x16 (x,y) patch of the MRI.
	The z co-ordinate will be in blockIdx.y
	The (x,y) location of the patch will be derived from blockIdx.x.
	Since the slices array is padded out to multiples of 16 by
	the other MRI slices routines (the host routine checks this), we
	don't have to worry about nasty edge cases
      */
      // Extract some co-ordinates
      const unsigned int by = blockIdx.x / (src.dims.x/kConv1dBlockSize);
      const unsigned int bx = blockIdx.x % (src.dims.x/kConv1dBlockSize);
      const unsigned int iz = blockIdx.y;
      const unsigned int tx = threadIdx.x;
      const unsigned int ty = threadIdx.y;
      
      const unsigned int ixStart = bx * kConv1dBlockSize;
      const unsigned int iyStart = by * kConv1dBlockSize;
      const unsigned int myx = ixStart + tx;
      const unsigned int myy = iyStart + ty;
      // Round up the convolution kernel size
      const unsigned int iConvKernelSize = MRIRoundUpConvolutionKernelSize();
      
      // Accumulator value
      float myVoxel = 0;
      
      // Declared as float because the GPU likes 4 byte datatypes
      __shared__ float srcCache[kConv1dBlockSize][kConv1dBlockSize];
      
      // Loop in the Y direction, loading sub-blocks of the src array into the cache
      // Note that yBegin can be negative, so we must cast iyStart and iConvKernelSize
      // to signed integers
      int yFirst = static_cast<int>(iyStart) - static_cast<int>(iConvKernelSize);
      int yLast = iyStart + iConvKernelSize;
      
      for( int yBegin = yFirst; yBegin <= yLast; yBegin += kConv1dBlockSize ) {
	// Load the cache
	srcCache[ty][tx] = src( myx, src.ClampCoord(yBegin+ty,src.dims.y), iz );
	__syncthreads();
	
	// Accumulate
	for( unsigned int i=0; i<kConv1dBlockSize; i++ ) {
	  int convLoc;
	  convLoc = (yBegin-iyStart) - static_cast<int>(ty);
	  convLoc += i;
	  convLoc += (dc_mriconv1d_kernel_nVals/2);
	  myVoxel += srcCache[i][tx] * GetMRIconvkernel( convLoc );
	}
	__syncthreads();
      }
      
      dst(myx, myy, iz) = GetFinal<U>(myVoxel);
      
    }
    



    //! Kernel to convolve in the Z direction
    template<typename T, typename U>
    __global__ void MRIConvolveKernelZ( const GPU::Classes::MRIframeOnGPU<T> src,
					GPU::Classes::MRIframeOnGPU<U> dst ) {
      /*!
	Kernel to do a convolution in the z direction, using the
	convolution kernel set up in the texture.
	Each block will do one 16x16 (x,z) patch of the MRI.
	The y co-ordinate will be in blockIdx.y
	The (x,z) location of the patch will be derived from blockIdx.x.
	Since the slices array is padded out to multiples of 16 by
	the other MRI slices routines (the host routine checks this), we
	don't have to worry about nasty edge cases
      */
      // Extract some co-ordinates
      const unsigned int bz = blockIdx.x / (src.dims.x/kConv1dBlockSize);
      const unsigned int bx = blockIdx.x % (src.dims.x/kConv1dBlockSize);
      const unsigned int iy = blockIdx.y;
      const unsigned int tx = threadIdx.x;
      // Note that we assign y thread index to tz, for naming ease
      const unsigned int tz = threadIdx.y;
      
      const unsigned int ixStart = bx * kConv1dBlockSize;
      const unsigned int izStart = bz * kConv1dBlockSize;
      const unsigned int myx = ixStart + tx;
      const unsigned int myz = izStart + tz;
      // Round up the convolution kernel size
      const unsigned int iConvKernelSize = MRIRoundUpConvolutionKernelSize();
      
      // Accumulator value
      float myVoxel = 0;

      // Declared as float because the GPU likes 4 byte datatypes
      __shared__ float srcCache[kConv1dBlockSize][kConv1dBlockSize];
      
      // Loop in the z direction, loading sub-blocks of the src array into the cache
      // Note that zBegin can be negative, so we must cast izStart and iConvKernelSize
      // to signed integers
      int zFirst = static_cast<int>(izStart) - static_cast<int>(iConvKernelSize);
      int zLast = izStart + iConvKernelSize;
      
      for( int zBegin = zFirst; zBegin <= zLast; zBegin += kConv1dBlockSize ) {
	// Load the cache
	srcCache[tz][tx] = src( myx, iy, src.ClampCoord(zBegin+tz,src.dims.z) );
	__syncthreads();
	
	// Accumulate
	for( unsigned int i=0; i<kConv1dBlockSize; i++ ) {
	  int convLoc;
	  convLoc = (zBegin-izStart) - static_cast<int>(tz);
	  convLoc += i;
	  convLoc += (dc_mriconv1d_kernel_nVals/2);
	  myVoxel += srcCache[i][tx] * GetMRIconvkernel( convLoc );
	}
	__syncthreads();
      }
      
      dst(myx, iy, myz) = GetFinal<U>(myVoxel);
  
    }






    //! Runs the 1D convolution kernel on the GPU
    template<typename T, typename U>
    void MRIConvolve1dGPU( const GPU::Classes::MRIframeGPU<T> &src,
			   GPU::Classes::MRIframeGPU<U> &dst,
			   const int axis,  
			   const hipStream_t myStream = 0 ) {
      
      /*!
	Runs the 1D convolution kernel on the GPU.
	Prior to calling this routine, MRIconv1d_SendKernel must
	be called to set up the convolution kernel
	@param[in] d_src The set of source frames
	@param[out] d_dst The set of destination frames
	@param[in] axis Which axis to do the convolution along
	@param[in] srcDims Dimensions of the source frames
	@param[in] dstDims Dimensions of the destinaiton frames
	@param[in] srcFrame Which frame of the source to use
	@param[in] dstFrame Which frame of the destination to use
	@param[in] myStream CUDA stream which should be used (Defaults to stream 0)
      */
      

      const dim3 srcDims = src.GetGPUDims();
      const dim3 dstDims = dst.GetGPUDims();

      // Check dimensions
      if( (srcDims.x != dstDims.x) &&
	  (srcDims.y != dstDims.y) &&
	  (srcDims.z != dstDims.z) ) {
	std::cerr << __FUNCTION__ << ": Dimension mismatch" << std::endl;
	exit( EXIT_FAILURE );
      }
      
      // Check padding (only need to do one, given above check)
      if( !dst.CheckPadding( kConv1dBlockSize ) ) {
	std::cerr << __FUNCTION__ <<
	  ": Arrays on GPU must be padded to multiples of kConv1dBlockSize" <<
	  std::endl;
	exit( EXIT_FAILURE );
      }
      
      
      dim3 grid, threads;
      GPU::Classes::MRIframeOnGPU<T> srcGPU(src);
      GPU::Classes::MRIframeOnGPU<U> dstGPU(dst);
      
      //cudaPrintfInit( 1024L*1024L*512 );
      
      switch( axis ) {
      case MRI_WIDTH:
	grid.x = (srcDims.x/kConv1dBlockSize) * (srcDims.y/kConv1dBlockSize);
	grid.y = srcDims.z;
	grid.z = 1;
	threads.x = threads.y = kConv1dBlockSize;
	threads.z = 1;
	
	MRIConvolveKernelX<T,U><<<grid,threads,0,myStream>>>( srcGPU, dstGPU );
	CUDA_CHECK_ERROR_ASYNC( "MRIconvolveKernelX failed!" );
	break;
	
      case MRI_HEIGHT:
	grid.x = (srcDims.x/kConv1dBlockSize) * (srcDims.y/kConv1dBlockSize);
	grid.y = srcDims.z;
	grid.z = 1;
	threads.x = threads.y = kConv1dBlockSize;
	threads.z = 1;
	
	MRIConvolveKernelY<T,U><<<grid,threads,0,myStream>>>( srcGPU, dstGPU );
	CUDA_CHECK_ERROR_ASYNC( "MRIconvolveKernelY failed!" );
	break;
	
      case MRI_DEPTH:
	// Slight change, since we do (x,z) patches
	grid.x = (srcDims.x/kConv1dBlockSize) * (srcDims.z/kConv1dBlockSize);
	grid.y = srcDims.y;
	threads.x = threads.y = kConv1dBlockSize;
	threads.z = 1;
	
	MRIConvolveKernelZ<T,U><<<grid,threads,0,myStream>>>( srcGPU, dstGPU );
	CUDA_CHECK_ERROR_ASYNC( "MRIconvolveKernelZ failed!" );
	break;
	
      default:
	std::cerr << __FUNCTION__ << ": Incompatible universe detected." << std::endl;
	std::cerr << "GPU functions are only tested ";
	std::cerr << "in a universe with three spatial dimensions" << std::endl;
	std::cerr << "Please adjust your reality accordingly, ";
	std::cerr << "and try again" << std::endl;
	exit( EXIT_FAILURE );
      }
      
      //cudaPrintfDisplay( stdout, true );
      
      //cudaPrintfEnd();
    }


    
    //! Dispatch routine with transfers
    template<typename T, typename U>
    void MRIConv1dDispatch( const MRI* src, MRI* dst,
			    const int axis,
			    const int srcFrame, const int dstFrame ) {
      /*!
	This is a dispatch routine for the 1D convolution on the GPU.
	It transfers the data to the GPU, runs the convolution, and retrieves
	the results
	Things are written this way to avoid nastily nested switch statements.
      */

      tMRIconv1dTotal.Start();

      GPU::Classes::MRIframeGPU<T> srcGPU;
      GPU::Classes::MRIframeGPU<U> dstGPU;
      
      char* h_workspace;
      size_t srcWorkSize, dstWorkSize;
      
      // Allocate the GPU arrays
      tMRIconv1dMem.Start();
      srcGPU.Allocate( src, kConv1dBlockSize );
      dstGPU.Allocate( dst, kConv1dBlockSize );
      tMRIconv1dMem.Stop();
      
      // Put in some sanity checks
      srcGPU.VerifyMRI( src );
      dstGPU.VerifyMRI( dst );
      
      // Allocate the workspace array
      tMRIconv1dMemHost.Start();
      srcWorkSize = srcGPU.GetBufferSize();
      dstWorkSize = dstGPU.GetBufferSize();
      
      if( srcWorkSize > dstWorkSize ) {
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_workspace,
				   srcWorkSize,
				       hipHostMallocDefault ) );
      } else {
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_workspace,
				       dstWorkSize,
				       hipHostMallocDefault ) );
      }
      tMRIconv1dMemHost.Stop();

      // Send the source data
      tMRIconv1dSend.Start();
      srcGPU.Send( src, srcFrame, h_workspace );
      tMRIconv1dSend.Stop();
      
      // Run the convolution
      tMRIconv1dCompute.Start();
      MRIConvolve1dGPU( srcGPU, dstGPU, axis );
      tMRIconv1dCompute.Stop();
  
  // Retrieve the answers
      tMRIconv1dRecv.Start();
      dstGPU.Recv( dst, dstFrame, h_workspace );
      tMRIconv1dRecv.Stop();
      
      tMRIconv1dMemHost.Start();
      CUDA_SAFE_CALL( hipHostFree( h_workspace ) );
      tMRIconv1dMemHost.Stop();
      
      CUDA_CHECK_ERROR( "1D Convolution failure" );
      
      tMRIconv1dTotal.Stop();
      
      // No need to release - destructor will do that automatically
    }
    
    //! Dispatch wrapper
    template<typename T>
    void MRIConv1dDispatchWrap( const MRI* src, MRI* dst,
				const int axis,
				const int srcFrame, const int dstFrame ) {
      switch( dst->type ) {
      case MRI_UCHAR:
	MRIConv1dDispatch<T,unsigned char>( src, dst, axis, srcFrame, dstFrame );
	break;
	
      case MRI_SHORT:
	MRIConv1dDispatch<T,short>( src, dst, axis, srcFrame, dstFrame );
	break;
	
      case MRI_FLOAT:
	MRIConv1dDispatch<T,float>( src, dst, axis, srcFrame, dstFrame );
	break;

      default:
	std::cerr << __FUNCTION__ << ": Unrecognised destination MRI type " << dst->type << std::endl;
	exit( EXIT_FAILURE );
      }
    }
    
  }
}

// =================================================


MRI* MRIconvolve1d_cuda( const MRI* src, MRI* dst,
			 const float *kernel, const int kernelLength,
			 const int axis,
			 const int srcFrame, const int dstFrame ) {
  /*!
    Reimplementation of MRIconvolve1d for the GPU.
    This is the 'drop in' replacement, and so has to do a lot of
    data transfers.
    As such, I don't expect it to be fast
  */


  // Get the convolution kernel to the GPU
  GPU::Algorithms::MRIconv1d_SendKernel( kernel, kernelLength );

  switch( src->type ) {
  case MRI_UCHAR:
    GPU::Algorithms::MRIConv1dDispatchWrap<unsigned char>( src, dst, axis, srcFrame, dstFrame );
    break;

  case MRI_SHORT:
    GPU::Algorithms::MRIConv1dDispatchWrap<short>( src, dst, axis, srcFrame, dstFrame );
    break;

  case MRI_FLOAT:
    GPU::Algorithms::MRIConv1dDispatchWrap<float>( src, dst, axis, srcFrame, dstFrame );
    break;
    
  default:
    std::cerr << __FUNCTION__ << ": Unrecognised source MRI type " << src->type << std::endl;
    exit( EXIT_FAILURE );
  }
  
  // Release the convolution kernel
  GPU::Algorithms::MRIconv1d_ReleaseKernel();


  return( dst );
}



// ======================================================

//! Stream insertion operator for timer
static std::ostream& operator<<( std::ostream& os,
				 const SciGPU::Utilities::Chronometer& timer ) {
  
  os << std::setw(9) << std::setprecision(6) << timer.GetAverageTime() << " ms (avg) ";
  os << std::setw(9) << std::setprecision(6) << timer.GetTime() << " ms (tot)";

  return( os );
}

void MRIconvShowTimers( void ) {
  /*!
    Pretty prints timers to std.out
  */

  std::cout << "=============================================" << std::endl;
  std::cout << "GPU convolution timers" << std::endl;
  std::cout << "----------------------" << std::endl;
#ifndef CUDA_FORCE_SYNC
  std::cout << "WARNING: CUDA_FORCE_SYNC not #defined" << std::endl;
  std::cout << "Timings may not be accurate" << std::endl;
#endif
  std::cout << std::endl;

  std::cout << "MRIConv1dDispatch" << std::endl;
  std::cout << "Host Memory   : " << GPU::Algorithms::tMRIconv1dMemHost << std::endl;
  std::cout << "GPU Memory    : " << GPU::Algorithms::tMRIconv1dMem << std::endl;
  std::cout << "Send          : " << GPU::Algorithms::tMRIconv1dSend << std::endl;
  std::cout << "Compute       : " << GPU::Algorithms::tMRIconv1dCompute << std::endl;
  std::cout << "Receive       : " << GPU::Algorithms::tMRIconv1dRecv << std::endl;
  std::cout << "------------------------------------------" << std::endl;
  std::cout << "Total : " << GPU::Algorithms::tMRIconv1dTotal << std::endl;
  std::cout << std::endl;

  std::cout << "=============================================" << std::endl;
}

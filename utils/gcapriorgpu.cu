/**
 * @file  gcapriorgpu.hpp
 * @brief Class to hold a volume of GCA priors in linear memory on the GPU
 *
 */
/*
 * Original Authors: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2011/01/25 15:55:32 $
 *    $Revision: 1.1 $
 *
 * Copyright (C) 2002-2010,
 * The General Hospital Corporation (Boston, MA).
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include "cudacheck.h"

#include "gcapriorgpu.hpp"


namespace GPU {
  namespace Classes {

    void GCApriorGPU::Allocate( const long long nxDim,
				const long long nyDim,
				const long long nzDim,
				const size_t num4D ) {
      // Get rid of old allocation
      this->Release();

      // Copy sizes
      this->xDim = nxDim;
      this->yDim = nyDim;
      this->zDim = nzDim;
      
      this->n4D = num4D;

      // Do the allocation
      const size_t nVoxels = this->xDim * this->yDim * this->zDim;

      // The offset array
      CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_offsets4D),
				  (nVoxels+1)*sizeof(size_t) ) );

      // Space for maxLabels
      CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_maxLabels),
				  nVoxels*sizeof(short) ) );

      // Space for the labels
      CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_labels),
				  this->n4D*sizeof(unsigned short) ) );

      // Space for the priors
      CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_priors),
				  this->n4D*sizeof(float) ) );

      // Space for the total_training
      CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_totTraining),
				  nVoxels*sizeof(int) ) );
    }


    // --------------------

    void GCApriorGPU::Release( void ) {

      if( this->xDim != 0 ) {
	// Release offset array
	CUDA_SAFE_CALL( hipFree( this->d_offsets4D ) );
	this->d_offsets4D = NULL;

	// Release 3D arrays
	CUDA_SAFE_CALL( hipFree( this->d_maxLabels ) );
	this->d_maxLabels = NULL;
	CUDA_SAFE_CALL( hipFree( this->d_totTraining ) );
	this->d_totTraining = NULL;

	// Release 4D arrays
	CUDA_SAFE_CALL( hipFree( this->d_labels ) );
	this->d_labels = NULL;
	CUDA_SAFE_CALL( hipFree( this->d_priors ) );
	this->d_priors = NULL;

	// Zero sizes
	this->xDim = 0;
	this->yDim = 0;
	this->zDim = 0;
	this->n4D = 0;
	
      }
    }
    

    // --------------------

    void GCApriorGPU::Send( const Freesurfer::GCAlinearPrior& src ) {

      // Allocate memory
      this->Allocate( src.xDim, src.yDim, src.zDim, src.n4D );

      const size_t nVoxels = this->xDim * this->yDim * this->zDim;

      // Copy offsets array
      CUDA_SAFE_CALL( hipMemcpy( this->d_offsets4D,
				  &src.offsets4D.front(),
				  (nVoxels+1)*sizeof(size_t),
				  hipMemcpyHostToDevice ) );

      // Copy 3D arrays
      CUDA_SAFE_CALL( hipMemcpy( this->d_maxLabels,
				  &src.maxLabels.front(),
				  nVoxels*sizeof(short),
				  hipMemcpyHostToDevice ) );
      CUDA_SAFE_CALL( hipMemcpy( this->d_totTraining,
				  &src.totTraining.front(),
				  nVoxels*sizeof(int),
				  hipMemcpyHostToDevice ) );

      // Copy 4D arrays
      CUDA_SAFE_CALL( hipMemcpy( this->d_labels,
				  &src.labels.front(),
				  this->n4D*sizeof(unsigned short),
				  hipMemcpyHostToDevice ) );
      CUDA_SAFE_CALL( hipMemcpy( this->d_priors,
				  &src.priors.front(),
				  this->n4D*sizeof(float),
				  hipMemcpyHostToDevice ) );
    }

  }
}

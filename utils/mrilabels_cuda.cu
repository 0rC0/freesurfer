#include "hip/hip_runtime.h"
/**
 * @file  mrilabels_cuda.cu
 * @brief Holds various MRI 'label' routines for the GPU
 *
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2011/01/05 20:15:36 $
 *    $Revision: 1.2 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 * Bug reports: analysis-bugs@nmr.mgh.harvard.edu
 *
 */
#include <cstdlib>
#include <cstdio>

#include <iostream>
#include <iomanip>

#include "mri.h"

#include "chronometer.hpp"
#include "cudacheck.h"
#include "mriframegpu.hpp"

#include "mrilabels_cuda.hpp"

// ==============================================


//! Texture for unsigned char source
texture<unsigned char, 3, hipReadModeElementType> dt_src_uchar;

//! Texture for unsigned char mri
texture<unsigned char, 3, hipReadModeElementType> dt_mri_uchar;

//! Texture for unsigned char mri_vals
texture<unsigned char, 3, hipReadModeElementType> dt_mri_vals_uchar;


namespace GPU {
  namespace Algorithms {

    //! Helper function to get texture values
    __device__
    unsigned char FetchSrcVoxel( const int ix,
				 const int iy,
				 const int iz ) {
      unsigned char texVal;

      texVal = tex3D( dt_src_uchar, ix+0.5f, iy+0.5f, iz+0.5f );

      return( texVal );
    }


    
    //! GPU kernel for MarkLabelBorderVoxel
    template<bool sixConnect>
    __global__
    void MarkLabelBorderVoxelKernel( GPU::Classes::MRIframeOnGPU<unsigned char> dst,
				     const int label,
				     const int mark ) {
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;
      
      // Loop over z
      for( unsigned int iz=0; iz<dst.dims.z; iz++ ) {

	// Ensure we're in range
	if( dst.InVolume(ix,iy,iz) ) {
	  const int this_label = FetchSrcVoxel( ix, iy, iz );
	  int border = 0;

	  // Loop over local volume
	  for( int xk=-1; xk<=1 && !border; xk++ ) {
	    for( int yk=-1; yk<=1 && !border; yk++ ) {
	      for( int zk=-1; zk<=1; zk++ ) {

		if( sixConnect && (abs(xk)+abs(yk)+abs(zk) != 1) ) {
		  continue;
		}

		const int that_label = FetchSrcVoxel( ix+xk, iy+yk, iz+zk );
		if( ((this_label == label) && (that_label != label)) ||
		    ((this_label != label) && (that_label == label)) ) {
		  border = 1 ;
		  break ;
		}
	      }
	    }
	  }
	  if( border ) {
	    dst(ix,iy,iz) = mark;
	  } else {
	    dst(ix,iy,iz) = 0;
	  }
	}
      }
    }
    

    // =============================================

    void MRIlabels::MarkLabelBorderVoxels( const GPU::Classes::MRIframeGPU<unsigned char>& src,
					   GPU::Classes::MRIframeGPU<unsigned char>& dst,
					   const int label,
					   const int mark,
					   const int sixConnect ) const {
      hipArray* srcArr = NULL;

      MRIlabels::tMarkLabelBorderVoxelsTot.Start();

      // Verify we have output allocated
      dst.Allocate( src );

      // Send the input to a cuda Array
      srcArr = src.CreateArray();

      // Bind the array to the texture
      dt_src_uchar.normalized = false;
      dt_src_uchar.addressMode[0] = hipAddressModeClamp;
      dt_src_uchar.addressMode[1] = hipAddressModeClamp;
      dt_src_uchar.addressMode[2] = hipAddressModeClamp;
      dt_src_uchar.filterMode = hipFilterModePoint;

      CUDA_SAFE_CALL( hipBindTextureToArray( dt_src_uchar, srcArr ) );

      // Run the kernel
      const unsigned int kKernelSize = 16;

      dim3 grid, threads;
      threads.x = threads.y = kKernelSize;
      threads.z = 1;

      grid = src.CoverBlocks( kKernelSize );
      grid.z = 1;

      if( sixConnect ) {
	MarkLabelBorderVoxelKernel<true><<<grid,threads>>>( dst, label, mark );
      } else {
	MarkLabelBorderVoxelKernel<false><<<grid,threads>>>( dst, label, mark );
      }
      CUDA_CHECK_ERROR( "MarkLabelBorderVoxelKernel failed!\n" );


      // Release the texture
      CUDA_SAFE_CALL( hipUnbindTexture( dt_src_uchar ) );

      // Release the hipArray
      CUDA_SAFE_CALL( hipFreeArray( srcArr ) );

      MRIlabels::tMarkLabelBorderVoxelsTot.Stop();
    }



    // ==============================================================

    template<typename T, unsigned int nVals>
    __device__
    void ZeroArray( T* arr ) {
      for( unsigned int i=0; i<nVals; i++ ) {
	arr[i] = 0;
      }
    }

    //! Texture wrapper for mri
    __device__
    unsigned char FetchMRIVoxel( const int ix,
				 const int iy,
				 const int iz ) {

      unsigned char texVal;

      texVal = tex3D( dt_mri_uchar, ix+0.5f, iy+0.5f, iz+0.5f );

      return( texVal );
    }


    //! Texture wrapper for mri_vals
    __device__
    unsigned char FetchMRIval( const int ix,
			       const int iy,
			       const int iz ) {
      unsigned char texVal;
      texVal = tex3D( dt_mri_vals_uchar, ix+0.5f, iy+0.5f, iz+0.5f );

      return( texVal );
    }

    
    //! Implementation of MRIcomputeLabelNbhd
    template<unsigned int nVals>
    __device__
    void ComputeLabelNbhd( const int x, const int y, const int z,
			   int *label_counts, float *label_means,
			   const int whalf ) {
      /*!
	This is an implementation of MRIcomputeLabelNbhd specific
	to the VoxInLabelPartVolumeKernel
	The two input MRIs (always assumed present) are passed
	via textures, since the same two are always supplied in
	MRIvoxelsInLabelWithPartialVolumeEffects.
	The size of the arrays is passed via the template
	parameter nVals
      */
      
      ZeroArray<int,nVals>( label_counts );
      ZeroArray<float,nVals>( label_means );

      for( int zk=-whalf; zk<=whalf; zk++ ) {
	for( int yk=-whalf; yk<=whalf; yk++ ) {
	  for( int xk=-whalf; xk<=whalf; xk++ ) {
	    const int label = FetchMRIVoxel( x+xk, y+yk, z+zk );
	    label_counts[label]++;

	    const float val = FetchMRIval( x+xk, y+yk, z+zk );
	    label_means[label] += val;
	  }
	}
      }

      for( int label=0; label<nVals; label++ ) {
	if( label_counts[label] > 0 ) {
	  label_means[label] /= label_counts[label];
	}
      }
    }


    template<unsigned int nVals>
    __global__
    void VoxInLabelPartVolumeKernel( const GPU::Classes::MRIframeOnGPU<unsigned char> mri_border,
				     GPU::Classes::MRIframeOnGPU<unsigned char> mri_nbr_labels,
				     GPU::Classes::MRIframeOnGPU<float> mri_mixing_coef,
				     const float vox_vol,
				     const int label,
				     float* volume ) {
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;


      // Loop over z
      for( unsigned int iz=0; iz<mri_nbr_labels.dims.z; iz++ ) {

	

	// Ensure we're in range
	if( mri_nbr_labels.InVolume(ix,iy,iz) ) {

	  const int vox_label = FetchMRIVoxel(ix,iy,iz);
	  const int border = mri_border(ix,iy,iz);
 
	  if( (vox_label!=label) && (border==0) ) {
	    continue;
	  }

	  if( border == 0 ) {
	    atomicAdd( volume, vox_vol );
	  } else {
	    int nbr_label_counts[nVals];
	    int label_counts[nVals];
	    float label_means[nVals];

	    ComputeLabelNbhd<nVals>( ix, iy, iz, nbr_label_counts, label_means, 1 );
	    ComputeLabelNbhd<nVals>( ix, iy, iz, label_counts, label_means, 7 );

	    const float val = FetchMRIval( ix, iy, iz );

	    const float mean_label = label_means[vox_label];
	    int nbr_label = -1;
	    int max_count = 0;
	    float pv, mean_nbr;

	    /*
	      look for a label that is a nbr and is
	      on the other side of val from the label mean
	    */
	    for( int this_label=0; this_label<nVals; this_label++ ) {

	      if( this_label == vox_label ) {
		continue ;
	      }

	      if( nbr_label_counts[this_label] == 0 ) {
		continue ; /* not a nbr */
	      }

	      if( (label_counts[this_label] > max_count) &&
		  ((label_means[this_label] - val) *
		   (mean_label - val) < 0) ) {
		max_count = label_means[this_label] ;
		nbr_label = this_label ;
	      }
	    }

	    if( vox_label != label && nbr_label != label ) {
	      continue; // this struct not in voxel 
	    }
	    

	    if( max_count == 0 ) {
	      atomicAdd( volume, vox_vol ); // couldn't find an appropriate label
	      
	      // find max nbr label anyway for caller
	      for( int this_label=0; this_label<nVals;  this_label++ ) {
		  
		if( this_label == vox_label ) {
		  continue;
		}
		
		if( nbr_label_counts[this_label] == 0 ) {
		  continue ; /* not a nbr */
		}
		  
		if( label_counts[this_label] > max_count ) {
		  max_count = label_means[this_label] ;
		  nbr_label = this_label ;
		}
	      }
		
	      mri_nbr_labels( ix, iy, iz ) = nbr_label;
	      mri_mixing_coef( ix, iy, iz ) = 1;	
	      
	      
	    } else {
	      // compute partial volume pct 
	      mean_nbr = label_means[nbr_label] ;
	      pv = (val - mean_nbr) / (mean_label - mean_nbr) ;
	      
	      if (pv > 1) {
		pv = 1 ;
	      }
	      
	      if (pv < 0) {
		continue ;  // shouldn't happen
	      }
	      
	      if( vox_label != label ) {
		pv = 1-pv ;
	      }
	      
	      atomicAdd( volume, vox_vol * pv );
	      
	      mri_mixing_coef( ix, iy, iz ) = pv;
	      
	      
	      if (vox_label != label) {
		mri_nbr_labels( ix, iy, iz ) = vox_label;
	      } else {
		mri_nbr_labels( ix, iy, iz ) = nbr_label;
	      }
	      
	      
	    }
	    
	  }
	}
      }
    }








    // ===========================

    //! GPU implementation of MRIvoxelsInLabelWithPartialVolumeEffects
    float MRIlabels::VoxInLabelWithPartialVolume( const GPU::Classes::MRIframeGPU<unsigned char>& mri,
						  const GPU::Classes::MRIframeGPU<unsigned char>& mri_vals,
						  const int label,
						  const int maxLabels,
						  GPU::Classes::MRIframeGPU<float>& mri_mixing_coeff,
						  GPU::Classes::MRIframeGPU<unsigned char>& mri_nbr_labels ) const {
      /*!
	This GPU implementation of MRIvoxelsInLabelWithPartialVolumeEffects
	assumes that both input MRIs (mri and mri_vals) are of type
	unsigned char.
      */
      
      MRIlabels::tVoxInLabelPartVolumeTot.Start();

      // Allocate  and zero the 'volume' global
      float *d_volume, h_volume;
      CUDA_SAFE_CALL( hipMalloc( (void**)&(d_volume), sizeof(float) ) );
      CUDA_SAFE_CALL( hipMemset( d_volume, 0, sizeof(float) ) );

      // Set up vox_vol
      const float3 sizes = mri.GetSizes();
      const float vox_vol = sizes.x*sizes.y*sizes.z;

      // Allocate the output MRIs
      mri_mixing_coeff.Allocate( mri );
      mri_mixing_coeff.Zero();
      mri_nbr_labels.Allocate( mri );
      mri_nbr_labels.Zero();

      // Declare the 'border' MRI
      GPU::Classes::MRIframeGPU<unsigned char> mriBorder;


      // Call the mark label border method
      this->MarkLabelBorderVoxels( mri, mriBorder, label, 1, 1 );


      // Set up the cuda Arrays
      hipArray* mriArr, *mri_valsArr;
      mriArr = mri_valsArr = NULL;

      mriArr = mri.CreateArray();
      mri_valsArr = mri_vals.CreateArray();

      
      // Set up the textures
      dt_mri_uchar.normalized = false;
      dt_mri_uchar.addressMode[0] = hipAddressModeClamp;
      dt_mri_uchar.addressMode[1] = hipAddressModeClamp;
      dt_mri_uchar.addressMode[2] = hipAddressModeClamp;
      dt_mri_uchar.filterMode = hipFilterModePoint;
      CUDA_SAFE_CALL( hipBindTextureToArray( dt_mri_uchar, mriArr ) );

      dt_mri_vals_uchar.normalized = false;
      dt_mri_vals_uchar.addressMode[0] = hipAddressModeClamp;
      dt_mri_vals_uchar.addressMode[1] = hipAddressModeClamp;
      dt_mri_vals_uchar.addressMode[2] = hipAddressModeClamp;
      dt_mri_vals_uchar.filterMode = hipFilterModePoint;
      CUDA_SAFE_CALL( hipBindTextureToArray( dt_mri_vals_uchar, mri_valsArr ) );

      
      // Run the computation
      const unsigned int kKernelSize = 16;

      dim3 grid, threads;
      threads.x = threads.y = kKernelSize;
      threads.z = 1;

      grid = mri.CoverBlocks( kKernelSize );
      grid.z = 1;
      MRIlabels::tVoxInLabelPartVolumeCompute.Start();
      if( maxLabels <= 1024 ) {
	VoxInLabelPartVolumeKernel<1024><<<grid,threads>>>( mriBorder,
							    mri_nbr_labels,
							    mri_mixing_coeff,
							    vox_vol,
							    label,
							    d_volume );
      } else {
	VoxInLabelPartVolumeKernel<32768><<<grid,threads>>>( mriBorder,
							     mri_nbr_labels,
							     mri_mixing_coeff,
							     vox_vol,
							     label,
							     d_volume );
      }
      CUDA_CHECK_ERROR( "VoxInLabelPartVolumeKernel failed!\n" );
      MRIlabels::tVoxInLabelPartVolumeCompute.Stop();

      // Unbind textures
      CUDA_SAFE_CALL( hipUnbindTexture( dt_mri_uchar ) );
      CUDA_SAFE_CALL( hipUnbindTexture( dt_mri_vals_uchar ) );

      // Release cudaArrays
      CUDA_SAFE_CALL( hipFreeArray( mriArr ) );
      CUDA_SAFE_CALL( hipFreeArray( mri_valsArr ) );


      // Retrieve the volume global and release
      CUDA_SAFE_CALL( hipMemcpy( &h_volume, d_volume,
				  sizeof(float),
				  hipMemcpyDeviceToHost ) );
      CUDA_SAFE_CALL( hipFree( d_volume ) );

      MRIlabels::tVoxInLabelPartVolumeTot.Stop();

      return( h_volume );
    }



    // ==============================================================

    // Declare statics
    SciGPU::Utilities::Chronometer MRIlabels::tMarkLabelBorderVoxelsTot;

    SciGPU::Utilities::Chronometer MRIlabels::tVoxInLabelPartVolumeTot;
    SciGPU::Utilities::Chronometer MRIlabels::tVoxInLabelPartVolumeCompute;


    void MRIlabels::ShowTimings( void ) {
      std::cout << "=============================================" << std::endl;
      std::cout << "GPU MRI Label timers" << std::endl;
      std::cout << "--------------------" << std::endl;
#ifndef CUDA_FORCE_SYNC
      std::cout << "WARNING: CUDA_FORCE_SYNC not #defined" << std::endl;
      std::cout << "Timings may not be accurate" << std::endl;
#endif
      std::cout << std::endl;

      std::cout << "MarkLabelBorderVoxels" << std::endl;
      std::cout << "Total : " << MRIlabels::tMarkLabelBorderVoxelsTot << std::endl;

      std::cout << "VoxInLabelWithPartialVolume" << std::endl;
      std::cout << "  Compute : " << MRIlabels::tVoxInLabelPartVolumeCompute << std::endl;
      std::cout << "Total       : " << MRIlabels::tVoxInLabelPartVolumeTot << std::endl;

      std::cout << "=============================================" << std::endl;

    }

  }
}

// ===============================================

static GPU::Algorithms::MRIlabels myLabels;

//! Wrapper for MRImarkLabelBorderVoxels
void MRImarkLabelBorderVoxelsGPU( const MRI* mri_src,
				  MRI* mri_dst,
				  int label,
				  int mark,
				  int six_connected ) {
  GPU::Classes::MRIframeGPU<unsigned char> srcGPU;
  GPU::Classes::MRIframeGPU<unsigned char> dstGPU;

  srcGPU.Allocate( mri_src );
  srcGPU.VerifyMRI( mri_src );

  srcGPU.SendFrame( mri_src, 0 );

  myLabels.MarkLabelBorderVoxels( srcGPU, dstGPU, label, mark, six_connected );

  dstGPU.RecvFrame( mri_dst, 0 );
}



//! Wrapper for MRIvoxelsInLabelWithPartialVolumeEffects

float MRIvoxelsInLabelWithPartialVolumeEffectsGPU( const MRI *mri,
						   const MRI *mri_vals, 
						   const int label,
						   const int maxlabels,
						   MRI *mri_mixing_coef, 
						   MRI *mri_nbr_labels ) {
  
  GPU::Classes::MRIframeGPU<unsigned char> mriGPU, mri_valsGPU;
  GPU::Classes::MRIframeGPU<unsigned char> mri_nbr_labelsGPU;
  GPU::Classes::MRIframeGPU<float> mri_mixing_coefGPU;

  // Send data to GPU
  mriGPU.Allocate( mri );
  mriGPU.VerifyMRI( mri );
  mriGPU.Send( mri, 0 );

  mri_valsGPU.Allocate( mri_vals );
  mri_valsGPU.VerifyMRI( mri_vals );
  mri_valsGPU.Send( mri_vals, 0 );

  // Run computation
  float vol = myLabels.VoxInLabelWithPartialVolume( mriGPU, mri_valsGPU,
						    label, maxlabels,
						    mri_mixing_coefGPU,
						    mri_nbr_labelsGPU );

  // Retrieve results
  if( mri_mixing_coef ) {
    mri_mixing_coefGPU.RecvFrame( mri_mixing_coef, 0 );
  }
  if( mri_nbr_labels ) {
    mri_nbr_labelsGPU.RecvFrame( mri_nbr_labels, 0 );
  }

  return( vol );
}

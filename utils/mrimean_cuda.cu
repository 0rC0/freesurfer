#include "hip/hip_runtime.h"
/**
 * @file  mrimean_cuda.cu
 * @brief Holds MRI mean function for the GPU
 *
 * Implements MRI mean function on the GPU
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: rge21 $
 *    $Date: 2010/02/17 20:49:34 $
 *    $Revision: 1.18 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 * Bug reports: analysis-bugs@nmr.mgh.harvard.edu
 *
 */

#include <cstdlib>
#include <cstdio>

#include <iostream>
#include <iomanip>


#include "chronometer.hpp"
#include "cudacheck.h"


#include "mriframegpu.hpp"


#include "mrimean_cuda.h"






namespace GPU {
  namespace Algorithms {

    const unsigned int kMRImeanBlockSize = 16;

    //! Min function for ints
    __device__ int min( const int& a, const int& b ) {
      if( a < b ) {
	return( a );
      } else { 
	return( b );
      }
    }

    //! Max function for ints
    __device__ int max( const int& a, const int& b ) {
      if( a > b ) {
	return( a );
      } else { 
	return( b );
      }
    }

    
    //! Kernel to compute x direction means
    template<typename T>
    __global__ void MRImeanX( const GPU::Classes::MRIframeOnGPU<T> src,
			      GPU::Classes::MRIframeOnGPU<float> dst,
			      const dim3 coverGrid,
			      const unsigned int wSize ) {
      /*!
	Kernel to compute means in the x direction, based on
	the given window size.
	Basically, does a 1D convolution, but with different
	boundary conditions to MRIConvolveKernelX.
	Also, since this is meant to be part of a pipeline,
	the destination type must be float
      */
      const unsigned int by = blockIdx.x / coverGrid.x;
      const unsigned int bx = blockIdx.x % coverGrid.x;
      const unsigned int tx = threadIdx.x;
      const unsigned int ty = threadIdx.y;
      
      const int ixStart = bx * kMRImeanBlockSize;
      const int iyStart = by * kMRImeanBlockSize;

      const int ix = ixStart + tx;
      const int iy = iyStart + ty;
      const int iz = blockIdx.y;

      const int wHalf = wSize/2;

      // Calculate voxels which will contribute, clamping to edges
      const unsigned int myxMin = max( 0           , ix - wHalf );
      const unsigned int myxMax = min( dst.dims.x-1, ix + wHalf );

      // Again, declare int to remove need for some casts
      const int patchSize = NextMultiple( max(wHalf,1), kMRImeanBlockSize );

      // Accumulator
      float myVal = 0;

      __shared__ float currPatch[kMRImeanBlockSize][kMRImeanBlockSize];

      // Calculate patch limits (note integer declarations avoid -ve trouble)
      const int xDimRound = NextMultiple( src.dims.x, kMRImeanBlockSize );
      const int xFirst = max( 0, ixStart - patchSize );
      const int xLast  = min( xDimRound - kMRImeanBlockSize,
			      ixStart + patchSize );

      for( int xBegin = xFirst; xBegin <= xLast; xBegin += kMRImeanBlockSize ) {
	// Load the patch
	currPatch[ty][tx] = src( xBegin+tx, iy, iz );
	__syncthreads();

	// Accumulate desired values
	for( unsigned int i=0; i<kMRImeanBlockSize; i++ ) {
	  int actx = xBegin + i;

	  if( (actx>=myxMin) && (actx<=myxMax) ) {
	    myVal += currPatch[ty][i];
	  }

	}

	__syncthreads();
      }

      // Save result
      if( dst.InVolume( ix, iy, iz ) ) {
	dst(ix,iy,iz) = dst.ConvertFloat( myVal );
      }
    }

    
    //! Kernel to compute y direction means
    template<typename T>
    __global__ void MRImeanY( const GPU::Classes::MRIframeOnGPU<T> src,
			      GPU::Classes::MRIframeOnGPU<float> dst,
			      const dim3 coverGrid,
			      const unsigned int wSize ) {
      /*!
	Kernel to compute means in the y direction, based on
	the given window size.
	Basically, does a 1D convolution, but with different
	boundary conditions to MRIConvolveKernelY.
	Also, since this is meant to be part of a pipeline,
	the destination type must be float
      */
      const unsigned int by = blockIdx.x / coverGrid.x;
      const unsigned int bx = blockIdx.x % coverGrid.x;
      const unsigned int tx = threadIdx.x;
      const unsigned int ty = threadIdx.y;
      
      const int ixStart = bx * kMRImeanBlockSize;
      const int iyStart = by * kMRImeanBlockSize;

      const int ix = ixStart + tx;
      const int iy = iyStart + ty;
      const int iz = blockIdx.y;

      const int wHalf = wSize/2;

      // Calculate voxels which will contribute, clamping to edges
      const unsigned int myyMin = max( 0           , iy - wHalf );
      const unsigned int myyMax = min( dst.dims.y-1, iy + wHalf );

      // Again, declare int to remove need for some casts
      const int patchSize = NextMultiple( max(wHalf,1), kMRImeanBlockSize );

      // Accumulator
      float myVal = 0;

      __shared__ float currPatch[kMRImeanBlockSize][kMRImeanBlockSize];

      // Calculate patch limits (note integer declarations avoid -ve trouble)
      const int yDimRound = NextMultiple( src.dims.y, kMRImeanBlockSize );

      const int yFirst = max( 0, iyStart - patchSize );
      const int yLast  = min( yDimRound - kMRImeanBlockSize,
			      iyStart + patchSize );

      for( int yBegin = yFirst; yBegin <= yLast; yBegin += kMRImeanBlockSize ) {
	// Load the patch
	currPatch[ty][tx] = src( ix, yBegin+ty, iz );
	__syncthreads();

	// Accumulate desired values
	for( unsigned int i=0; i<kMRImeanBlockSize; i++ ) {
	  int acty = yBegin + i;

	  if( (acty>=myyMin) && (acty<=myyMax) ) {
	    myVal += currPatch[i][tx];
	  }

	}

	__syncthreads();
      }

      // Save result
      if( dst.InVolume( ix, iy, iz ) ) {
	dst(ix,iy,iz) = dst.ConvertFloat( myVal );
      }
    }


    //! Kernel to compute z direction means
    template<typename T>
    __global__ void MRImeanZ( const GPU::Classes::MRIframeOnGPU<T> src,
			      GPU::Classes::MRIframeOnGPU<float> dst,
			      const dim3 coverGrid,
			      const unsigned int wSize ) {
      /*!
	Kernel to compute means in the x direction, based on
	the given window size.
	Basically, does a 1D convolution, but with different
	boundary conditions to MRIConvolveKernelZ.
	Also, since this is meant to be part of a pipeline,
	the destination type must be float
      */
      const unsigned int bz = blockIdx.x / coverGrid.x;
      const unsigned int bx = blockIdx.x % coverGrid.x;
      const unsigned int tx = threadIdx.x;
      // Note... tz is threadIdx.y
      const unsigned int tz = threadIdx.y;
      
      const int ixStart = bx * kMRImeanBlockSize;
      const int izStart = bz * kMRImeanBlockSize;

      const int ix = ixStart + tx;
      const int iy = blockIdx.y;
      const int iz = izStart + tz;

      const int wHalf = wSize/2;

      // Calculate voxels which will contribute, clamping to edges
      const unsigned int myzMin = max( 0           , iz - wHalf );
      const unsigned int myzMax = min( dst.dims.z-1, iz + wHalf );

      // Again, declare int to remove need for some casts
      const int patchSize = NextMultiple( max(wHalf,1), kMRImeanBlockSize );

      // Accumulator
      float myVal = 0;

      __shared__ float currPatch[kMRImeanBlockSize][kMRImeanBlockSize];

      // Calculate patch limits (note integer declarations avoid -ve trouble)
      const int zDimRound = NextMultiple( src.dims.z, kMRImeanBlockSize );

      const int zFirst = max( 0, izStart - patchSize );
      const int zLast  = min( zDimRound - kMRImeanBlockSize,
			      izStart + patchSize );
 
      for( int zBegin = zFirst; zBegin <= zLast; zBegin += kMRImeanBlockSize ) {
	// Load the patch
	currPatch[tz][tx] = src( ix, iy, zBegin+tz );
	__syncthreads();

	// Accumulate desired values
	for( unsigned int i=0; i<kMRImeanBlockSize; i++ ) {
	  int actz = zBegin + i;

	  if( (actz>=myzMin) && (actz<=myzMax) ) {
	    myVal += currPatch[i][tx];
	  }

	}

	__syncthreads();
      }

      // Save result
      if( dst.InVolume( ix, iy, iz ) ) {
	dst(ix,iy,iz) = dst.ConvertFloat( myVal );
      }
    }


    //! Kernel to normalise means
    template<typename U>
    __global__ void MRImeanNormal( const GPU::Classes::MRIframeOnGPU<float> src,
				   GPU::Classes::MRIframeOnGPU<U> dst,
				   const dim3 coverGrid,
				   const unsigned int wSize ) {
      /*!
	Kernel to normalise the means computed by the earlier
	stages.
	As such, the input type must be a float
      */
      const unsigned int by = blockIdx.x / coverGrid.x;
      const unsigned int bx = blockIdx.x % coverGrid.x;
      const unsigned int tx = threadIdx.x;
      const unsigned int ty = threadIdx.y;
      
      const int ixStart = bx * kMRImeanBlockSize;
      const int iyStart = by * kMRImeanBlockSize;

      const int ix = ixStart + tx;
      const int iy = iyStart + ty;
      const int iz = blockIdx.y;

      const int wHalf = wSize/2;

      // Calculate voxels which contributed, clamping to edges
      const unsigned int myxMin = max( 0           , ix - wHalf );
      const unsigned int myxMax = min( dst.dims.x-1, ix + wHalf );
      const unsigned int myyMin = max( 0           , iy - wHalf );
      const unsigned int myyMax = min( dst.dims.y-1, iy + wHalf );
      const unsigned int myzMin = max( 0           , iz - wHalf );
      const unsigned int myzMax = min( dst.dims.z-1, iz + wHalf );


      const unsigned long myVolume = ( myxMax - myxMin + 1 ) *
	(myyMax - myyMin + 1 ) *
	(myzMax - myzMin + 1 );

      if( dst.InVolume( ix, iy, iz ) ) {
	dst( ix, iy, iz ) = dst.ConvertFloat( src( ix, iy, iz ) / myVolume );
      }
    }



    


    // ######################################################

    //! Wrapper class for the MRI mean algorithm
    class MRImean {
    private:
      //! Stream which should be used for this instance
      hipStream_t stream;
      //! Private pinned memory workspace
      mutable char* h_workspace;
      //! Size of private workspace
      mutable size_t workSize;

      mutable SciGPU::Utilities::Chronometer tMem, tHostMem;
      mutable SciGPU::Utilities::Chronometer tSend, tRecv, tCompute;
      mutable SciGPU::Utilities::Chronometer tTotal;
      
      // =======================
      
      // Wrapper function
      template<typename T>
      void DispatchWrap( const MRI* src, MRI* dst,
			 const unsigned int wSize,
			 const int srcFrame, const int dstFrame ) const {
	switch( dst->type ) {
	case MRI_UCHAR:
	  this->MeanDispatch<T,unsigned char>( src, dst, wSize, srcFrame, dstFrame );
	  break;

	case MRI_SHORT:
	  this->MeanDispatch<T,short>( src, dst, wSize, srcFrame, dstFrame );
	  break;
	  
	case MRI_FLOAT:
	  this->MeanDispatch<T,float>( src, dst, wSize, srcFrame, dstFrame );
	  break;

	default:
	  std::cerr << __FUNCTION__
		    << ": Unrecognised destination MRI type "
		    << dst->type
		    << std::endl;
	  exit( EXIT_FAILURE );
	}
      }
      
      // =========================

      //! Ensures internal pinned memory buffer is at least of size nBytes
      void Allocate( const size_t nBytes ) const {
	if( this->workSize < nBytes ) {
	  this->Release();

	  this->tHostMem.Start();
	  CUDA_SAFE_CALL( hipHostAlloc( (void**)&(this->h_workspace),
					 nBytes,
					 hipHostMallocDefault ) );
	  this->workSize = nBytes;
	  this->tHostMem.Stop();
	}
      }
	  

      //! Releases internal pinned memory buffer
      void Release( void ) const {
	if( h_workspace != NULL ) {
	  this->tHostMem.Start();
	  CUDA_SAFE_CALL( hipHostFree( h_workspace ) );
	  h_workspace = NULL;
	  workSize = 0;
	  this->tHostMem.Stop();
	}
      }

      // =========================
      // Prevent copying

      MRImean( const MRImean& src ) : stream(0),
				      h_workspace(NULL),
				      workSize(0),
				      tMem(), tHostMem(),
				      tSend(), tRecv(),
				      tCompute(), tTotal() {
	std::cerr << __FUNCTION__
		  << ": Please don't copy these objects"
		  << std::endl;
	exit( EXIT_FAILURE );
      }

      MRImean& operator=( const MRImean& src ) {
	std::cerr << __FUNCTION__
		  << ": Please don't copy these objects"
		  << std::endl;
	exit( EXIT_FAILURE );
      }


    public:
      //! Constructor with stream (also default constructor)
      MRImean( const hipStream_t s = 0 ) : stream(s),
					    h_workspace(NULL),
					    workSize(0),
					    tMem(), tHostMem(),
					    tSend(), tRecv(),
					    tCompute(), tTotal() {}
      
      //! Destructor
      ~MRImean( void ) {
	this->Release();
#ifdef CUDA_SHOW_TIMINGS
	std::cout << "==================================" << std::endl;
	std::cout << "GPU Mean timers" << std::endl;
	std::cout << "---------------" << std::endl;
#ifndef CUDA_FORCE_SYNC
	std::cout << "WARNING: CUDA_FORCE_SYNC not #defined" << std::endl;
	std::cout << "Timings may not be accurate" << std::endl;
#endif
	std::cout << std::endl;

	std::cout << "Host Memory : " << this->tHostMem << std::endl;
	std::cout << "GPU Memory  : " << this->tMem << std::endl;
	std::cout << "Send        : " << this->tSend << std::endl;
	std::cout << "Compute     : " << this->tCompute << std::endl;
	std::cout << "Receive     : " << this->tRecv << std::endl;
	std::cout << "--------------" << std::endl;
	std::cout << "Total : " << this->tTotal << std::endl;
	std::cout << "==================================" << std::endl;
#endif
      }


      //! Dispatch for data on the CPU of unknown type
      void DoMean( const MRI* src, MRI* dst,
		   const unsigned int wSize,
		   const unsigned int srcFrame = 0,
		   const unsigned int dstFrame = 0 ) const {
	switch( src->type ) {
	case MRI_UCHAR:
	  this->DispatchWrap<unsigned char>( src, dst, wSize, srcFrame, dstFrame );
	  break;
  
	case MRI_SHORT:
	  this->DispatchWrap<short>( src, dst, wSize, srcFrame, dstFrame );
	  break;
	  
	case MRI_FLOAT:
	  this->DispatchWrap<float>( src, dst, wSize, srcFrame, dstFrame );
	  break;
 
	default:
	  std::cerr << __FUNCTION__
		    << ": Unrecognised source MRI type "
		    << src->type
		    << std::endl;
	  exit( EXIT_FAILURE );
	}
      }
      

      //! Templated dispatch for known data types
      template<typename T, typename U>
      void MeanDispatch( const MRI* src, MRI* dst,
			 const unsigned int wSize,
			 const int srcFrame, const int dstFrame ) const {
	/*!
	  Templated dispatch routine for MRI mean function on the
	  GPU.
	  Transfers data to the GPU, and retrieves the results
	*/
	this->tTotal.Start();

	GPU::Classes::MRIframeGPU<T> srcGPU;
	GPU::Classes::MRIframeGPU<U> dstGPU;

	size_t srcWorkSize, dstWorkSize;
      
	// Allocate the GPU arrays
	this->tMem.Start();
	srcGPU.Allocate( src );
	dstGPU.Allocate( dst );
	this->tMem.Stop();

	// Put in some sanity checks
	srcGPU.VerifyMRI( src );
	dstGPU.VerifyMRI( dst );
      
	// Allocate the workspace array
	srcWorkSize = srcGPU.BufferSize();
	dstWorkSize = dstGPU.BufferSize();
      
	if( srcWorkSize > dstWorkSize ) {
	  this->Allocate( srcWorkSize );
	} else {
	  this->Allocate( dstWorkSize );
	}

	// Send the source data
	this->tSend.Start();
	srcGPU.Send( src, srcFrame, this->h_workspace, this->stream );
	this->tSend.Stop();

	// Run the filter
	this->RunGPU( srcGPU, dstGPU, wSize );

	// Get the results
	this->tRecv.Start();
	dstGPU.Recv( dst, dstFrame, this->h_workspace, this->stream );
	this->tRecv.Stop();

	CUDA_CHECK_ERROR( "Mean filtering failure" );

	this->tTotal.Stop();
      }


      
      //! Runs the mean filtering kernel
      template<typename T, typename U>
      void RunGPU( const GPU::Classes::MRIframeGPU<T> &src,
		   GPU::Classes::MRIframeGPU<U> &dst,
		   const unsigned int wSize ) const {
	/*!
	  Runs the mean filtering kernel on the GPU.
	  Assumes most things are properly set already
	*/
	const dim3 srcDims = src.GetDims();
	const dim3 dstDims = dst.GetDims();

	// Check dimensions
	if( srcDims != dstDims ) {
	  std::cerr << __FUNCTION__ << ": Dimension mismatch"
		    << std::endl;
	  exit( EXIT_FAILURE );
	}


	// We need intermediates which are floats
	GPU::Classes::MRIframeGPU<float> f1, f2;

	// Get correctly sized intermediates
	this->tMem.Start();
	f1.Allocate( src );
	f2.Allocate( src );
	this->tMem.Stop();

	// Create the GPU kernel objects
#if 0
	GPU::Classes::MRIframeOnGPU<T> srcGPU(src);
	GPU::Classes::MRIframeOnGPU<float> f1GPU( f1 );
	GPU::Classes::MRIframeOnGPU<float> f2GPU( f2 );
	GPU::Classes::MRIframeOnGPU<U> dstGPU(dst);
#endif


	// Do the three convolutions. Recall objects have same dims
	dim3 grid, threads;

	
	threads.x = threads.y = kMRImeanBlockSize;
	threads.z = 1;

	const dim3 coverGrid = dst.CoverBlocks( kMRImeanBlockSize );

	grid.x = coverGrid.x * coverGrid.y;
	grid.y = dstDims.z;
	grid.z = 1;

	this->tCompute.Start();

	// Do the X direction
	MRImeanX<T>
	  <<<grid,threads,0,this->stream>>>
	  ( src, f1, coverGrid, wSize );
	CUDA_CHECK_ERROR_ASYNC( "MRImeanX kernel failed" );

	// Do the Y direction
	MRImeanY<float>
	  <<<grid,threads,0,this->stream>>>
	  ( f1, f2, coverGrid, wSize );
	CUDA_CHECK_ERROR_ASYNC( "MRImeanY kernel failed" );

	// Slight change for Z direction
	grid.x = coverGrid.x * coverGrid.z;
	grid.y = dstDims.y;
	MRImeanZ<float>
	  <<<grid,threads,0,this->stream>>>
	  ( f2, f1, coverGrid, wSize );
	CUDA_CHECK_ERROR_ASYNC( "MRImeanZ kernel failed" );

	// Normalise
	grid.x = coverGrid.x * coverGrid.y;
	grid.y = dstDims.z;
	MRImeanNormal<U>
	  <<<grid,threads,0,this->stream>>>
	  ( f1, dst, coverGrid, wSize );
	CUDA_CHECK_ERROR_ASYNC( "MRImeanNormal failed!" );

	this->tCompute.Stop();
      }

    };

  }
}


static GPU::Algorithms::MRImean myMean;


MRI* MRImean_cuda( const MRI* src, MRI* dst,
		   const unsigned int wSize ) {
  /*!
    Wrapper around the GPU routine, to be called from the
    original MRImean routine.
    Note that the frames default to zero, per the original
    MRImean routine.
  */

  myMean.DoMean( src, dst, wSize );

  return( dst );
}
